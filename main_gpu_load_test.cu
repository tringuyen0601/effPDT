//#include  "processor.h"
#include "gpu_fst_engine.h"
#include <stdlib.h>
#include <stdint.h>
#include <iostream>
#include <cstring>

int main ( int argc, char *argv[]){

  uint32_t threads= 0;
  uint32_t blocks = 0;
  uint32_t size_32b = 0;
  float size_gb = 0;
  uint32_t mode = 0;
 
  std:: cout << " SET thread per block : " << THREAD_PER_BLOCK << std::endl; 
  std:: cout << " SET MODE: " << MODE << std::endl; 
  std:: cout << " SET " << STATE_COUNT << " state"  << std::endl; 
  std:: cout << " SET " << TRANSITION_COUNT<< " transition"  << std::endl; 
  std:: cout << " SET VARIABLE : " << VARIABLE << std::endl; 
  std:: cout << " SET INPUT : " << INPUT << std::endl; 
  std:: cout << " SET OUTPUT : " << OUTPUT << std::endl; 
  std::cout << "Reading: " << argc << " arggument" << std::endl;
  if ( argc != 6) {
    std::cout << " need 6 argument ( config file, number_of_block, number_of_threads, size in GB( accepting floating point), execution mode ), only get " << argc<< " argument"<< std::endl;
 	std::cout << " Mode 0: Global Memory" << std::endl;
 	std::cout << " Mode 1: Topology in Shared Memory" << std::endl;
  exit(0);
  }
  std::string configFile  = std::string(argv[1]);
  mode = atoi( argv[5]);
  size_gb = atof( argv[4]);
  size_32b = (uint32_t) ( (size_gb * 1e9 )/ 4);
  threads = atoi( argv[3]);
  blocks = atoi( argv[2]);
  std::cout  << "block: : " << blocks<<std::endl;
  std::cout  << "thread: : " << threads<<std::endl;
  std::cout  << "input Size: : " << size_gb<<"GB or " << size_32b<< " element of 32bit"  <<std::endl;
  std::cout << "Config File: "<< configFile <<std::endl; 
  std::cout <<" Setting up argument" << std::endl;
  ARGUMENT  * argm = new ARGUMENT  ( configFile, size_32b);
  std::cerr << " Input Size : " << size_gb <<" GB" <<std::endl; 
  FSTGPU * FST_ENGINE = (FSTGPU *)malloc(sizeof( FSTGPU));
  TP * processor  = new TP( argm->config);
  printf("LOAD FST TO CPU\n");
  processor->loadFST_CPU();
  printf("INITIALIZE GPU FST\n");
  TPGPU * Hprocessor;
  hipMalloc(&Hprocessor,  sizeof(TPGPU));
  
  printf("LOAD FST TO GPU\n");
  processor->loadFST_GPU( Hprocessor, blocks, threads);
  printf("START PRINTING FROM KERNEL\n");
  //FST_ENGINE->printFST( Hprocessor, 1,1,1);
  FST_ENGINE->IO_setup( processor, blocks, threads);
  FST_ENGINE->IO_partition (processor,  argm->benchmark_int);
  if ( mode == 0){
    FST_ENGINE->process (Hprocessor );
  }
  else if ( mode == 1){
    FST_ENGINE->process_shared(Hprocessor );
  }
  else if ( mode == 2){
    FST_ENGINE->topo_global_to_constant(processor);
    FST_ENGINE->process_constant(Hprocessor );
  }
  else if ( mode == 3){
    FST_ENGINE->topo_global_to_constant_thread(processor);
    FST_ENGINE->process_thread(Hprocessor );
  
  }
  std::cout <<" Application Finishes" << std::endl;
//return ;
}
