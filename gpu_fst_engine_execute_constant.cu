#include "hip/hip_runtime.h"
#include "processor_gpu.h"
#include "gpu_fst_engine.h"

//__constant__  NPDT stateList[STATE_COUNT];
//__constant__ Transition  transitionList[TRANSITION_COUNT];

__device__
void ExecuteAction_d_constant ( TPGPU * fst, uint32_t src1, uint32_t src2, uint32_t dst, uint32_t opt, uint32_t startVar){
  uint32_t src1Value, src2Value;
  switch (opt){
    case ADD:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value + src2Value;
      break;
    case ADDI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value + src2Value;
      break;
    case SUB:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value - src2Value;
      break;
    case SUBI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value - src2Value;
      break;
    case MUL:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value * src2Value;
      break;
    case MULI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value * src2Value;
      break;
    case DIV:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value / src2Value;
      break;
    case DIVI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value / src2Value;
      break;
    case LSHIFT:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value << src2Value;
      break;
    case LSHIFTI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value << src2Value;
      break;
    case RSHIFT:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value >> src2Value;
      break;
    case RSHIFTI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value >> src2Value;
      break;
    case OR:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value | src2Value;
      break;
    case ORI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value | src2Value;
      break;
    case AND:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = (fst->var[src2+ startVar]);
      fst->var[dst + startVar] = src1Value & src2Value;
      break;
    case ANDI:
      src1Value = (fst->var[src1 + startVar]);
      src2Value = src2;
      fst->var[dst + startVar] = src1Value & src2Value;
      break;
    case EPSILON:
    default:
      break;
  }
}


__device__
void state_action_d_constant ( TPGPU * fst, uint32_t  currentState, uint32_t startVar){
  uint32_t src1 = stateList[currentState].src1 ;
  uint32_t src2 = stateList[currentState].src2 ;
  uint32_t dst = stateList[currentState].dst ;
  uint32_t opt = stateList[currentState].opt;
  ExecuteAction_d_constant ( fst, src1, src2, dst, opt, startVar);
}


__device__
uint32_t getInput_d_constant ( uint32_t *  input, uint32_t * processed_symbol, uint32_t * input_base, uint32_t inputID ){
  uint32_t base = input_base[inputID];
  uint32_t current =  processed_symbol[inputID];
  return input[base+  current];

}

__device__
bool compareSwitch_d_constant( TPGPU* fst, uint32_t lhs, uint32_t rhs, uint32_t ** input, uint32_t *processed_symbol, uint32_t * input_base, uint32_t startVar){
  int condition = ILL;
  // alphabet match:  0-256
  if ( rhs < ALPHABETSIZE) {
    //printf(" alphabet match\n");
    condition = ALPHABET;
  }
  // negation alphabet  match: 256-511
  else if ( (rhs >= ALPHABETSIZE) && ( rhs < (2 * ALPHABETSIZE)) ) {
    condition = NEGALPHABET;
  }
  // EPSILON match: 512
  else if ( rhs == (2 * ALPHABETSIZE)) {
    condition = EPS;
  }
  // Variable match: 513-576
  else if (( rhs >  (2 * ALPHABETSIZE)) && ( rhs <= (2 * (ALPHABETSIZE) + MAXVAR))) {
    condition = VAR;
  }
  //  Input Match:  577-640
  else if( ( rhs >= INPUT_MATCH) && ( rhs <( INPUT_MATCH + MAXVAR))) {
    condition = INPT;
  }
  //  Passthrough Match:  641
  else if ( rhs == ANY_MATCH) {
    condition = PASSTHROUGH;
  }
  // negation variable match:  769-832
  else if (( rhs >  (3 * ALPHABETSIZE)) && ( rhs <= (3 * ALPHABETSIZE + MAXVAR))) {
    condition =NEGVAR;
  }
  // Negation Input Match:  833-896
  else if( ( rhs >=( INPUT_MATCH + ALPHABETSIZE)) && ( rhs <( INPUT_MATCH+ ALPHABETSIZE+ MAXVAR))) {
    condition = NEGINPT;
    }

  uint32_t varValue, inputValue;
  bool match = false;
  switch (condition)  {
    case  ALPHABET:
      if ( lhs == rhs) match = true;
      break;
    case  NEGALPHABET:
      if ( lhs  != (rhs- ALPHABETSIZE )) match = true;
      break;
    case  EPS:
      match = true;
      break;
    case  VAR:
      varValue = fst->var[rhs- 2*ALPHABETSIZE + startVar -1 ];
      if ( lhs == varValue) match = true;
      break;
    case  INPT:
      inputValue = getInput_d_constant( input[rhs-INPUT_MATCH], processed_symbol, input_base, rhs -INPUT_MATCH);
      if ( lhs == inputValue) match = true;
      break;
    case  PASSTHROUGH:
      match = true;
      break;
    case  NEGVAR:
      varValue = fst->var[rhs- 3*ALPHABETSIZE -1 + startVar];
      if ( lhs != varValue) match = true;
      break;
    case  NEGINPT:
      inputValue = getInput_d_constant( input[rhs-INPUT_MATCH- ALPHABETSIZE], processed_symbol, input_base, rhs - INPUT_MATCH - ALPHABETSIZE);
      if ( lhs != inputValue) match = true;
      break;
      }
    return match;

}



__device__
uint32_t transition_eval_d_constant(TPGPU * fst, uint32_t  currentState,  uint32_t startVar, uint32_t **input, uint32_t * input_base,  uint32_t *processed_symbol){
  uint32_t baseID = stateList[currentState].baseID;
  uint32_t numberofTransition = stateList[currentState].numberofTransition;
  uint32_t returnID = TRANSITION_COUNT;
  for ( uint32_t i = baseID; i < (baseID + numberofTransition); i++){
    // input
    bool inputMatch = false;
    if ( transitionList[i].inputID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lhs = getInput_d_constant(input[ transitionList[i].inputID- INPUT_MATCH], processed_symbol, input_base, transitionList[i].inputID- INPUT_MATCH);
      /*
      printf(" Transition %d :", i);
      printf(" left hand side = %d ;", lhs);
      printf(" right hand side = %d \n", fst->transitionList[i].inputSymbol);
      */
      // call compare to right-hand side
      inputMatch = compareSwitch_d_constant ( fst, lhs,transitionList[i].inputSymbol, input, processed_symbol, input_base,  startVar);
    }
    else {
      inputMatch = true;
    }
  // variable
  bool varMatch = false;
    if ( transitionList[i].IvarID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lshVar = transitionList[i].IvarID + startVar - 2 *ALPHABETSIZE -1;
      uint32_t lhs = fst->var[lshVar];
      // call compare to right-hand side
      varMatch = compareSwitch_d_constant ( fst, lhs,transitionList[i].inputVar , input, processed_symbol,  input_base, startVar);
    }
    else {
      varMatch = true;
    }

  if ( inputMatch && varMatch){
     return i;
    }
  }
  return returnID;
}


__device__
uint32_t translateOutput_d_constant ( TPGPU * fst, uint32_t ** input, uint32_t * processed_symbol, uint32_t * input_base,  uint32_t outputVar, uint32_t startVar){
  uint32_t character = 1000; // illegal value

  if ( outputVar < ALPHABETSIZE)
    character = outputVar;
  else if (( outputVar >=ALPHABETSIZE) && ( outputVar <= 2*ALPHABETSIZE))
    return character;
  else if ( (outputVar > 2*(ALPHABETSIZE)) &&( outputVar < (2*ALPHABETSIZE + MAXVAR)) ){
    character = fst->var[startVar + outputVar - 2*ALPHABETSIZE -1 ];

  }
  else if ( outputVar >= 2 * ALPHABETSIZE + MAXVAR +1){
    character = getInput_d_constant( input[outputVar- INPUT_MATCH], processed_symbol,input_base, outputVar - INPUT_MATCH);
    //printf("Write to output %d: %d\n", outputVar-INPUT_MATCH, character);
  }
  return character;
}



__device__
uint32_t transition_write_d_constant( TPGPU * fst, uint32_t chosenTransition, uint32_t startVar, uint32_t ** input,  uint32_t * input_base,  uint32_t*  processed_symbol, uint32_t ** output,uint32_t * simulated_output, uint32_t* outputCount , uint32_t *output_base){
  if( chosenTransition == TRANSITION_COUNT)
    return STATE_COUNT;

  // write to variable
  if ( transitionList[chosenTransition].OvarID != EPSILON_MATCH){
    uint32_t outputVar = startVar+ transitionList[chosenTransition].OvarID - 2 * ALPHABETSIZE - 1;
    //printf("Write Var %d \n", outputVar);
    fst->var[outputVar]  =
      translateOutput_d_constant ( fst, input, processed_symbol,  input_base, transitionList[chosenTransition].outputVar,startVar);
  }
  // write to output

  if ( transitionList[chosenTransition].outputID != EPSILON_MATCH){
    uint32_t output_dst = transitionList[chosenTransition].outputID - OUTPUT_START;
#ifdef DEBUG_GPU
    output[output_dst][output_base[output_dst] + outputCount[output_dst]]  =
#else
  simulated_output[output_dst]  =
#endif
      translateOutput_d_constant ( fst, input, processed_symbol, input_base, transitionList[chosenTransition].outputSymbol, startVar);
    outputCount[output_dst] ++;
  }

  //actiavate next State
  return transitionList[chosenTransition].nextState;
}



__device__
bool is_done_processed_constant ( uint32_t * current, uint32_t * expected, uint32_t number_of_input){
  for ( int i = 0; i < number_of_input; i++){
    //printf(" Current %d: %d| Expected %d\n", i,  current[i] , expected[i]);
    if ( current[i] < expected[i])
      return false;
  }
  return true;
}

//////////////////
// main processing function
// partition_length: number of inputsymbol each stream have to processed
// input_base: element that each stream have to processed from
// processed_symbol: number of input each stream have processed
// partition_output_base: element each stream have to process from
// output_count: number of symbol have been written to each stream
__global__
void process_k_constant( TPGPU * fst, uint32_t ** input_base, uint32_t ** partition_length, uint32_t ** processed_symbol, uint32_t ** input, 
                             uint32_t ** partition_output_base, uint32_t** partition_output_count, uint32_t ** output, uint32_t **simulated_output ){
  uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t total_thread =   gridDim.x * blockDim.x ;
  uint32_t startVar = globalId * VARIABLE;
  uint32_t currentState = 0;
  uint32_t chosenTransition = TRANSITION_COUNT;
  bool done = false;
  uint32_t cycle= 0;
  uint32_t t = 0;
#ifdef DEBUG_GPU
  if (globalId == 0){
    printf( "State Table : %d state\n", STATE_COUNT);
    for ( uint32_t i = 0 ; i < STATE_COUNT; i++)
      printf("S[%d]: %d, %d, %d, %d\n", stateList[i].id,  stateList[i].opt, stateList[i].src1, stateList[i].src2, stateList[i].dst);
    printf( "Transition Table : %d transition\n", TRANSITION_COUNT);
    for ( uint32_t i = 0 ; i < TRANSITION_COUNT; i++){
      printf("T[%d]: %d-%d, %d-%d, %d | ", transitionList[i].id, transitionList[i].inputID, transitionList[i].inputSymbol, transitionList[i].IvarID, transitionList[i].inputVar, transitionList[i].currentState);
      printf(" %d-%d, %d-%d, %d \n", transitionList[i].outputID, transitionList[i].outputSymbol, transitionList[i].OvarID, transitionList[i].outputVar, transitionList[i].nextState);
    }
    
  }
#endif


#ifdef DEBUG_GPU
  while ( cycle < 49){

    cycle++;
    __syncthreads();
if ( globalId == 0){
    printf("-------------%d-----------------\n", cycle);
    printf( " Activated State: %d\n", currentState);
    for ( uint32_t v = 0; v < fst->varCount; v++)
      printf( "V[%d] = %u | ", v, fst->var[v]);
    printf("\n");
    // print input

    for (uint32_t p = 0; p < fst->inputCount; p++){

      printf("Input[%d]: ", p);
      for (uint32_t q = 0; q < 10; q++){
        printf("%d, ", input[p][q]);
      }

      printf("\n");
      printf("Current[%d]: 0 - %d total %d \n",p,  processed_symbol[globalId][p], partition_length[globalId][p]);
      for (uint32_t k = 0; k <=  10; k++){
        printf("%u, ", input[p][k]);
      }
      printf("\n");
    }
    for (uint32_t p = 0; p < fst->outputCount; p++){
      printf("OutPut[%d]: ", p);
      for (uint32_t k = partition_output_base[globalId][p]; k < partition_output_base[globalId][p] + partition_output_count[globalId][p]; k++){
        printf("%u, ", output[p][k]);
      }
      printf("\n");
    }
    }
    __syncthreads();
    //printf("Rerform Action\n");
#else

  while (( !done) && (currentState != STATE_COUNT)){
#endif


    state_action_d_constant ( fst, currentState, startVar);
    // transition evaluate
    uint32_t baseID = stateList[currentState].baseID;
    uint32_t numberofTransition = stateList[currentState].numberofTransition;
    //printf(" Considering Transition %d to %d\n", baseID, baseID+ numberofTransition -1 );
    chosenTransition = transition_eval_d_constant( fst, currentState, startVar,  input,input_base[globalId],  processed_symbol[globalId]);
    // transition write
  currentState =
    transition_write_d_constant(fst, chosenTransition, startVar,input,  input_base[globalId], processed_symbol[globalId],  output,simulated_output[globalId],  partition_output_count[globalId], partition_output_base[globalId]);
    // consumed input and deactivate transition
    if (currentState == STATE_COUNT){
      done = true;
    }
    else {
      if( transitionList[chosenTransition].inputID != EPSILON_MATCH) {
        processed_symbol[globalId][transitionList[chosenTransition].inputID-INPUT_MATCH]++;
        chosenTransition = TRANSITION_COUNT;
        }
    done = is_done_processed_constant( processed_symbol[globalId], partition_length[globalId], INPUT);
    }
  }
  __syncthreads();
#ifdef DEBUG_GPU
  if ( globalId == 0)
    printf("Thread %d finished with %d cycle, %d output \n",  globalId, cycle, partition_output_count[globalId][0]);
    printf( " total Thread %d \n",  gridDim.x * blockDim.x );
    if ( globalId == 0){
      for ( uint32_t i = 0; i <(blockDim.x * gridDim.x) ; i++){
        for ( uint32_t j = 0; j < fst->inputCount; j++){
          printf("T%d:  %d, ",i, partition_output_base[i][j]);
          printf(" %d|  ", partition_output_count[i][j]);
        }
      }
    }
#endif

}

void FSTGPU::process_constant( TPGPU * transducer){
  printf("Begin Execution\n");
  printf(" Number of block: %d \n", number_of_block);
  printf(" Number of thread per block: %d\n", number_of_thread);
  hipEvent_t start_execution, stop_execution;
  hipEventCreate( &start_execution);
  hipEventCreate( &stop_execution);
  hipEventRecord( start_execution, 0);
  process_k_constant <<< number_of_block,number_of_thread>>> (transducer, partition_input_base, partition_input_length, partition_input_current, input,  partition_output_base,  partition_output_current,  output , simulated_output);
  hipEventRecord( stop_execution,0);
  hipEventSynchronize(stop_execution);
  float execution_time_ms = 0;
  hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
  //hipDeviceSynchronize();
  printf("Execution  Finished\n");
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess )
    printf("CUDA Error: %s\n", hipGetErrorString(err));

  fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}

void FSTGPU::topo_global_to_constant(TP * cpu_transducer){

  hipMemcpyToSymbol(HIP_SYMBOL( stateList), cpu_transducer->stateList, STATE_COUNT * sizeof(NPDT));
  hipMemcpyToSymbol(HIP_SYMBOL( transitionList), cpu_transducer->transitionList, TRANSITION_COUNT * sizeof(Transition));
} 
