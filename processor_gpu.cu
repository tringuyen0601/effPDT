#include "processor.h"

void TP::loadFST_GPU( TPGPU * h_processor, uint32_t blocks, uint32_t threads){
  /* stack and memory info  */
  hipError_t error; 
  printf( "START LOADING \n");
  bool * temp_bool; 
  uint32_t * temp ;
  NPDT * temp_npdt;
  Transition * temp_trans;

  error = hipMalloc(&(temp) ,varCount  * blocks * threads * sizeof(uint32_t));
  error = hipMemcpy(&h_processor->var, &temp, sizeof(uint32_t*), hipMemcpyHostToDevice);
  hipMemset(temp, 0, varCount * blocks * threads * sizeof( uint32_t));

  error = hipMalloc(&(temp_npdt) ,stateCount * sizeof(NPDT));
  error = hipMemcpy(&h_processor->stateList, &temp_npdt, sizeof(NPDT*), hipMemcpyHostToDevice);


  error = hipMalloc(&(temp_trans) ,transitionCount * sizeof(Transition));
  error = hipMemcpy(&h_processor->transitionList, &temp_trans, sizeof(Transition*), hipMemcpyHostToDevice);
  error = hipMalloc(&(temp_bool) ,transitionCount * sizeof(bool));
  error = hipMemcpy(&h_processor->pendingTransition, &temp_bool, sizeof(bool*), hipMemcpyHostToDevice);



  printf( "START COPY counter \n");
  error = hipMemcpy(&h_processor->stateCount, &stateCount, sizeof(uint32_t), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "GPU assert%s \n", hipGetErrorString(error));
  }
  error = hipMemcpy(&h_processor->transitionCount, &transitionCount, sizeof(uint32_t), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "GPU assert%s \n", hipGetErrorString(error));
  }
  error = hipMemcpy(&h_processor->varCount, &varCount, sizeof(uint32_t), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "GPU assert%s \n", hipGetErrorString(error));
  }
  error = hipMemcpy(&h_processor->inputCount, &inputCount, sizeof(uint32_t), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "GPU assert%s \n", hipGetErrorString(error));
  }
  error = hipMemcpy(&h_processor->outputCount,  & outputCount, sizeof(uint32_t), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "GPU assert%s \n", hipGetErrorString(error));
  }


  printf( "START COPY Topology \n");
  /*
  for ( int i = 0; i < varCount; i ++)
    printf( "var[%d] = %d\n", var[i], i);
  */
  if ( error != hipSuccess){
    printf( "var GPU assert %s \n", hipGetErrorString(error));
  }
  error = hipMemcpy(temp_npdt, stateList, stateCount * sizeof(NPDT), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "state GPU assert %s \n", hipGetErrorString(error));
  }
  error = hipMemcpy(temp_trans,transitionList,  transitionCount * sizeof(Transition), hipMemcpyHostToDevice);
  if ( error != hipSuccess){
    printf( "transition GPU assert %s \n", hipGetErrorString(error));
  }
}
