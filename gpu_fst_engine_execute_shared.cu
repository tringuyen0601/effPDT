#include "hip/hip_runtime.h"
#include "processor_gpu.h"
#include "gpu_fst_engine.h"

__device__
void ExecuteAction_d_shared ( uint32_t * var, uint32_t src1, uint32_t src2, uint32_t dst, uint32_t opt, uint32_t startVar){
  uint32_t src1Value, src2Value;
  switch (opt){
    case ADD:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value + src2Value;
      break;
    case ADDI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value + src2Value;
      break;
    case SUB:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value - src2Value;
      break;
    case SUBI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value - src2Value;
      break;
    case MUL:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value * src2Value;
      break;
    case MULI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value * src2Value;
      break;
    case DIV:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value / src2Value;
      break;
    case DIVI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value / src2Value;
      break;
    case LSHIFT:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value << src2Value;
      break;
    case LSHIFTI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value << src2Value;
      break;
    case RSHIFT:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value >> src2Value;
      break;
    case RSHIFTI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value >> src2Value;
      break;
    case OR:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value | src2Value;
      break;
    case ORI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value | src2Value;
      break;
    case AND:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value & src2Value;
      break;
    case ANDI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value & src2Value;
      break;
    case EPSILON:
    default:
      break;
  }
}

__device__
void state_action_d_shared ( TPGPU * fst, uint32_t * var, uint32_t  currentState, uint32_t startVar){
  uint32_t src1 = fst->stateList[currentState].src1 ;
  uint32_t src2 = fst->stateList[currentState].src2 ;
  uint32_t dst = fst->stateList[currentState].dst ;
  uint32_t opt = fst->stateList[currentState].opt;
  ExecuteAction_d_shared ( var, src1, src2, dst, opt, startVar);
}


__device__
uint32_t getInput_d_shared ( uint32_t *  input, uint32_t * processed_symbol, uint32_t * input_base, uint32_t inputID ){
  uint32_t base = input_base[inputID];
  uint32_t current =  processed_symbol[inputID];
  return input[base+  current];

}


__device__
bool compareSwitch_d( uint32_t * var, uint32_t lhs, uint32_t rhs, uint32_t ** input, uint32_t * shared_input_base,  uint32_t *shared_processed_symbol,  uint32_t startVar){

  int condition = ILL;
  // alphabet match:  0-256
  if ( rhs < ALPHABETSIZE) {
    //printf(" alphabet match\n");
    condition = ALPHABET;
  }
  // negation alphabet  match: 256-511
  else if ( (rhs >= ALPHABETSIZE) && ( rhs < (2 * ALPHABETSIZE)) ) {
    condition = NEGALPHABET;
  }
  // EPSILON match: 512
  else if ( rhs == (2 * ALPHABETSIZE)) {
    condition = EPS;
  }
  // Variable match: 513-576
  else if (( rhs >  (2 * ALPHABETSIZE)) && ( rhs <= (2 * (ALPHABETSIZE) + MAXVAR))) {
    condition = VAR;
  }
  //  Input Match:  577-640
  else if( ( rhs >= INPUT_MATCH) && ( rhs <( INPUT_MATCH + MAXVAR))) {
    condition = INPT;
  }
  //  Passthrough Match:  641
  else if ( rhs == ANY_MATCH) {
    condition = PASSTHROUGH;
  }
  // negation variable match:  769-832
  else if (( rhs >  (3 * ALPHABETSIZE)) && ( rhs <= (3 * ALPHABETSIZE + MAXVAR))) {
    condition =NEGVAR;
  }
  // Negation Input Match:  833-896
  else if( ( rhs >=( INPUT_MATCH + ALPHABETSIZE)) && ( rhs <( INPUT_MATCH+ ALPHABETSIZE+ MAXVAR))) {
    condition = NEGINPT;
    }

  uint32_t varValue, inputValue;
  bool match = false;
  switch (condition)  {
    case  ALPHABET:
      if ( lhs == rhs) match = true;
      break;
    case  NEGALPHABET:
      if ( lhs  != (rhs- ALPHABETSIZE )) match = true;
      break;
    case  EPS:
      match = true;
      break;
    case  VAR:
      varValue = var[rhs- 2*ALPHABETSIZE + startVar -1 ];
      if ( lhs == varValue) match = true;
      break;
    case  INPT:
      inputValue = getInput_d_shared( input[rhs-INPUT_MATCH], shared_processed_symbol, shared_input_base,  rhs -INPUT_MATCH);

      if ( lhs == inputValue) match = true;
      break;
    case  PASSTHROUGH:
      match = true;
      break;
    case  NEGVAR:
      varValue = var[rhs- 3*ALPHABETSIZE -1 + startVar];
      if ( lhs != varValue) match = true;
      break;
    case  NEGINPT:
      inputValue = getInput_d_shared( input[rhs-INPUT_MATCH- ALPHABETSIZE], shared_processed_symbol, shared_input_base,  rhs - INPUT_MATCH - ALPHABETSIZE);
      if ( lhs != inputValue) match = true;
      break;
      }
    return match;

}


__device__
uint32_t transition_eval_d_shared(TPGPU * fst, uint32_t * var, uint32_t  currentState,  uint32_t startVar, uint32_t **input, uint32_t * shared_input_base,   uint32_t *shared_processed_symbol){
  uint32_t baseID = fst->stateList[currentState].baseID;
  uint32_t numberofTransition = fst->stateList[currentState].numberofTransition;
  uint32_t returnID = fst->transitionCount;
  for ( uint32_t i = baseID; i < (baseID + numberofTransition); i++){
    // input
    bool inputMatch = false;
    if ( fst->transitionList[i].inputID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lhs = getInput_d_shared(input[ fst->transitionList[i].inputID- INPUT_MATCH], shared_processed_symbol, shared_input_base,  fst->transitionList[i].inputID- INPUT_MATCH);
      /*
      printf(" Transition %d :", i);
      printf(" left hand side = %d ;", lhs);
      printf(" right hand side = %d \n", fst->transitionList[i].inputSymbol);
      */
      // call compare to right-hand side
      inputMatch = compareSwitch_d ( var,  lhs,fst->transitionList[i].inputSymbol, input, shared_input_base,  shared_processed_symbol, startVar);

    }
    else {
      inputMatch = true;
    }
  // variable
  bool varMatch = false;
    if ( fst->transitionList[i].IvarID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lshVar = fst->transitionList[i].IvarID + startVar - 2 *ALPHABETSIZE -1;
      uint32_t lhs = var[lshVar];
      // call compare to right-hand side
      varMatch = compareSwitch_d ( var, lhs,fst->transitionList[i].inputVar , input, shared_input_base, shared_processed_symbol, startVar);

    }
    else {
      varMatch = true;
    }

  if ( inputMatch && varMatch){
      //printf(" Match transition %d\n", i);
     return i;
    }
  }
  return returnID;
}


__device__
uint32_t translateOutput_d ( uint32_t * var, uint32_t ** input, uint32_t * shared_input_base,  uint32_t * shared_processed_symbol,uint32_t outputVar, uint32_t startVar){

  uint32_t character = 1000; // illegal value

  if ( outputVar < ALPHABETSIZE)
    character = outputVar;
  else if (( outputVar >=ALPHABETSIZE) && ( outputVar <= 2*ALPHABETSIZE))
    return character;
  else if ( (outputVar > 2*(ALPHABETSIZE)) &&( outputVar < (2*ALPHABETSIZE + MAXVAR)) ){
    character = var[startVar + outputVar - 2*ALPHABETSIZE -1 ];

  }
  else if ( outputVar >= 2 * ALPHABETSIZE + MAXVAR +1){
    character = getInput_d_shared( input[outputVar- INPUT_MATCH], shared_processed_symbol,shared_input_base,  outputVar - INPUT_MATCH);

    //printf("Write to output %d: %d\n", outputVar-INPUT_MATCH, character);
  }
  return character;
}



__device__
uint32_t transition_write_d_shared( TPGPU * fst,  uint32_t * var, uint32_t chosenTransition, uint32_t startVar, uint32_t ** input, uint32_t * shared_input_base,  uint32_t*  processed_symbol, uint32_t ** output,uint32_t * simulated_output, uint32_t* outputCount , uint32_t *output_base){
  if( chosenTransition == fst->transitionCount)
    return fst->stateCount;

  // write to variable
  if ( fst->transitionList[chosenTransition].OvarID != EPSILON_MATCH){

    uint32_t outputVar = startVar+ fst->transitionList[chosenTransition].OvarID - 2 * ALPHABETSIZE - 1;
    //printf("Write Var %d \n", outputVar);
    var[outputVar]  =
      translateOutput_d ( var,  input, shared_input_base,  processed_symbol, fst->transitionList[chosenTransition].outputVar,startVar);
  }
  // write to output

  if ( fst->transitionList[chosenTransition].outputID != EPSILON_MATCH){
    uint32_t output_dst = fst->transitionList[chosenTransition].outputID - OUTPUT_START;
#ifdef DEBUG_GPU
    output[output_dst][output_base[output_dst] + outputCount[output_dst]]  =
#else
  simulated_output[output_dst]  =
#endif
      translateOutput_d ( var, input, shared_input_base, processed_symbol, fst->transitionList[chosenTransition].outputSymbol, startVar);

    outputCount[output_dst] ++;
   // printf("AFTER Write output  %d \n", tmp);
    //printf(" From %d ", output_base[output_dst]);
    //printf("To: %d\n",outputCount[output_dst]);

  }

  //actiavate next State
  return fst->transitionList[chosenTransition].nextState;
}


__device__
bool is_done_processed_shared ( uint32_t * current, uint32_t * expected, uint32_t number_of_input){
  for ( int i = 0; i < number_of_input; i++){
    if ( current[i] < expected[i])
      return false;
  }
  return true;
}


//////////////////
// main processing function
// partition_length: number of inputsymbol each stream have to processed
// input_base: element that each stream have to processed from
// processed_symbol: number of input each stream have processed
// partition_output_base: element each stream have to process from
// output_count: number of symbol have been written to each stream
__global__
void process_k_shared( TPGPU * fst,uint32_t ** input_base,  uint32_t ** partition_length, uint32_t ** processed_symbol, uint32_t ** input, 
                             uint32_t ** partition_output_base, uint32_t** partition_output_count, uint32_t ** output, uint32_t **simulated_output ){
  uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t total_thread =   gridDim.x * blockDim.x ;
  uint32_t startVar = globalId * VARIABLE;
  uint32_t startVar_shared = threadIdx.x * VARIABLE;

  __shared__ uint32_t var [VARIABLE * THREAD_PER_BLOCK];
  __shared__ uint32_t shared_input_base[THREAD_PER_BLOCK][INPUT];
  __shared__ uint32_t shared_input_length[THREAD_PER_BLOCK][INPUT]; 
  __shared__ uint32_t shared_input_current[THREAD_PER_BLOCK][INPUT]; 

// initialize shared variable
  for ( uint32_t i = 0; i < INPUT; i++){
    shared_input_base[threadIdx.x][i] = input_base[globalId][i];
    shared_input_length[threadIdx.x][i] = partition_length[globalId][i];
    shared_input_current[threadIdx.x][i] = 0;
  } 
  for ( uint32_t i = 0; i < VARIABLE; i++){
    var[threadIdx.x *VARIABLE + i] = 0 ;
  } 
  __syncthreads();
#ifdef DEBUG_GPU
  uint32_t chosen_thread = 0;
  if ( globalId == chosen_thread){
    for (uint32_t i = startVar_shared ; i < ( startVar_shared +  VARIABLE ) ; i++){
      printf( "V[%d]=%d , ", i , var[i]);
    }

    printf("\n");
      for (uint32_t j = 0; j < INPUT; j++){
        printf( "Input_base[%d][%d]=%d , ", threadIdx.x,   j, shared_input_base[threadIdx.x][j]);
        printf( "Input_length[%d][%d]=%d , ",threadIdx.x,  j, shared_input_length[threadIdx.x][j]);
        printf( "Input_current[%d][%d]=%d  ",threadIdx.x,j, shared_input_current[threadIdx.x][j]);
      }
    printf("\n");
  }
  __syncthreads();
#endif
// execution
  
  uint32_t currentState = 0;
  uint32_t stateCount = STATE_COUNT;
  bool done = false;
  uint32_t chosenTransition = TRANSITION_COUNT;
  uint32_t cycle = 0;
#ifdef DEBUG_GPU
  while ( cycle < 20){
    if ( globalId == chosen_thread){
      printf("-------------%d-----------------\n", cycle);
      printf( " Activated State: %d\n", currentState);
      for (uint32_t v = startVar_shared; v < ( startVar_shared+  VARIABLE ) ; v++)
        printf( "V[%d] = %u | ", v, var[v]);
      printf("\n");

      for (uint32_t p = 0; p < INPUT; p++){
        printf("Input[%d]: ", p);
      for (uint32_t q = shared_input_base[threadIdx.x][p]; q <(shared_input_base[threadIdx.x][p] +   10); q++)            printf("%d, ", input[p][q]);
      printf("\n");

      printf("Current[%d]:  %d - %d total %d \n",p,shared_input_base[threadIdx.x][p],  shared_input_current[threadIdx.x][p], shared_input_length[threadIdx.x][p]);      for (uint32_t k = shared_input_base[threadIdx.x][p]; k <(shared_input_base[globalId][p] +  shared_input_current[threadIdx.x][p]); k++)        printf("%u, ", input[p][k]);
      printf("\n");
      }
      for (uint32_t p = 0; p < OUTPUT; p++){
        printf("OutPut[%d]: ", p);
        for (uint32_t k = partition_output_base[globalId][p]; k < partition_output_base[globalId][p] + partition_output_count[globalId][p]; k++){          printf("%u, ", output[p][k]);
        }
        printf("\n");
      }
    printf("Perform Action\n");
    }
    cycle++;
    __syncthreads();
#else
  while (  ( !done) && ( currentState != stateCount)){
#endif

    // state action
    state_action_d_shared ( fst, var, currentState, threadIdx.x * VARIABLE);
#ifdef DEBUG_GPU
    if ( globalId == chosen_thread) {
      printf( "Perform Action on state %d, var %d\n", currentState, threadIdx.x * VARIABLE);
      uint32_t baseID = fst->stateList[currentState].baseID;
      uint32_t numberofTransition = fst->stateList[currentState].numberofTransition;
      printf(" Considering Transition %d to %d\n", baseID, baseID+ numberofTransition -1 );
    }
     __syncthreads();
#endif
    // transition evaluation

    chosenTransition = transition_eval_d_shared( fst, var, currentState, startVar_shared,  input,  shared_input_base[threadIdx.x], shared_input_current[threadIdx.x]);
#ifdef DEBUG_GPU
    if ( globalId == chosen_thread) {
      printf( "Chosen Transition  T%d\n", chosenTransition);
    }
     __syncthreads();
#endif

    currentState =
    transition_write_d_shared(fst, var, chosenTransition, startVar_shared,input,  shared_input_base[threadIdx.x], shared_input_current[threadIdx.x],  output,simulated_output[globalId],  partition_output_count[globalId], partition_output_base[globalId]);

#ifdef DEBUG_GPU
    if ( globalId == chosen_thread) printf( "Write Next State %d\n", currentState);
#endif


    // consumed input and deactivate transition
    if (currentState == fst->stateCount){
      done = true;
    }
    else {
      if( fst->transitionList[chosenTransition].inputID != EPSILON_MATCH) {
        shared_input_current[threadIdx.x][fst->transitionList[chosenTransition].inputID-INPUT_MATCH]++;
        chosenTransition = TRANSITION_COUNT;
        }
    done = is_done_processed_shared( shared_input_current[threadIdx.x], shared_input_length[threadIdx.x], fst->inputCount);
    }


  }
  __syncthreads();
#ifdef DEBUG_GPU
  if ( globalId == chosen_thread){
    printf("Thread %d finished with %d cycle, %d output \n",  globalId, cycle, partition_output_count[globalId][0]);
    printf( " total Thread %d \n",  gridDim.x * blockDim.x );
    for ( uint32_t j = 0; j < fst->inputCount; j++){
      printf("T%d:  %d, ",globalId,  partition_output_base[globalId][j]);
      printf(" %d|  ", partition_output_count[globalId][j]);
    }
  }
#endif


}

void FSTGPU::process_shared( TPGPU * transducer){
  printf("Begin Execution\n");
  printf(" Number of block: %d \n", number_of_block);
  printf(" Number of thread per block: %d\n", number_of_thread);
  hipEvent_t start_execution, stop_execution;
  hipEventCreate( &start_execution);
  hipEventCreate( &stop_execution);

  hipEventRecord( start_execution, 0);


  process_k_shared <<< number_of_block,number_of_thread>>> (transducer,  partition_input_base,  partition_input_length, partition_input_current, input,  partition_output_base,  partition_output_current,  output , simulated_output);
  hipEventRecord( stop_execution,0);
  hipEventSynchronize(stop_execution);
  float execution_time_ms = 0;
  hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
  //hipDeviceSynchronize();
  printf("Execution  Finished\n");
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess )
    printf("CUDA Error: %s\n", hipGetErrorString(err));       
  fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}
