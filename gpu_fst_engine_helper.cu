#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"

__global__ 
void printTransitionKernel1( TPGPU * transducer){
  printf("%d variable \n", transducer->varCount);
  printf("%d state \n", transducer->stateCount);
  printf("%d transition \n", transducer->transitionCount);
  for ( int i =0; i < transducer->varCount; i++){
    printf("variable[%d] = %d \n", i, transducer->var[i]);
  }

  for ( int i =0; i < transducer->stateCount; i++){
    printf("State %d \n", transducer->stateList[i].id);
  }
  for ( int i =0; i < transducer->transitionCount; i++){
    printf("Transition %d \n", transducer->transitionList[i].id);
  }
}

void FSTGPU::printFST( TPGPU * transducer, bool state, bool transition, bool variable){
  hipError_t error;

  TPGPU * TEMP_FST = (TPGPU *) malloc (sizeof( TPGPU)); 
  hipMemcpy( TEMP_FST, transducer, sizeof(TPGPU), hipMemcpyDeviceToHost);

  if (state){
    printf("============================================\n");
    printf( "%d States\n", TEMP_FST->stateCount);
    NPDT * stateList = (NPDT *) malloc (TEMP_FST->stateCount * sizeof(NPDT));
    error = hipMemcpy( stateList, TEMP_FST->stateList, TEMP_FST->stateCount * sizeof(NPDT), hipMemcpyDeviceToHost);

    if ( error != hipSuccess){
      printf( "GPU assert%s \n", hipGetErrorString(error));
    }
    for (int i = 0; i < TEMP_FST->stateCount; i++){
      printf("STATE %d ", stateList[i].id);
      action_convert ( stateList[i].opt,stateList[i].src1, stateList[i].src2, stateList[i].dst);
    }
  }
  
  if ( transition){
    printf("============================================\n");
    printf( "%d Transition\n", TEMP_FST->transitionCount);
    Transition * transitionList = (Transition *) malloc (TEMP_FST->transitionCount * sizeof(Transition));
    error = hipMemcpy( transitionList, TEMP_FST->transitionList, TEMP_FST->transitionCount * sizeof(Transition), hipMemcpyDeviceToHost);

    if ( error != hipSuccess){
      printf( "GPU assert%s \n", hipGetErrorString(error));
    }
    for (int i = 0; i < TEMP_FST->transitionCount; i++){
      printf("T%d :", transitionList[i].id);
      transition_convert ( transitionList[i].inputID);
      printf("==");
      transition_convert ( transitionList[i].inputSymbol);
      printf(",");
      transition_convert ( transitionList[i].IvarID);
      printf("==");
      transition_convert ( transitionList[i].inputVar);
      printf(" -> ");
      printf(" T%d ", transitionList[i].nextState);
      transition_convert ( transitionList[i].outputID);
      printf("==");
      transition_convert ( transitionList[i].outputSymbol);
      printf(",");
      transition_convert ( transitionList[i].OvarID);
      printf("==");
      transition_convert ( transitionList[i].outputVar);
      printf("\n");
    }
  }
  if( variable){
    printf("============================================\n");
    printf( "%d Variable\n", TEMP_FST->varCount);
    uint32_t * varList = (uint32_t *) malloc (TEMP_FST->varCount * sizeof(uint32_t));
    error = hipMemcpy( varList, TEMP_FST->var, TEMP_FST->varCount * sizeof(uint32_t), hipMemcpyDeviceToHost);

    if ( error != hipSuccess){
      printf( "GPU assert%s \n", hipGetErrorString(error));
    }
    for (int i = 0; i < TEMP_FST->varCount; i++){
      printf("S[%d] = %d \n", i,varList[i]);
    }
  }
}



void FSTGPU::action_convert( uint32_t opcode, uint32_t src1, uint32_t src2, uint32_t dst){
  switch ( opcode) {
    case 1: // ADD
      printf( "S[%d] = S[%d] + S[%d]\n", dst, src1, src2);;
      break;
    case 2: // ADD I
      printf( "S[%d] = S[%d] + %d\n", dst, src1, src2);;
      break;
    case 3: //  SUB
      printf( "S[%d] = S[%d] - S[%d]\n", dst, src1, src2);;
      break;
    case 4: // SUBI
      printf( "S[%d] = S[%d] - %d\n", dst, src1, src2);;
      break;
    case 5: //  MUL
      printf( "S[%d] = S[%d] * S[%d]\n", dst, src1, src2);;
      break;
    case 6: // MULI:
      printf( "S[%d] = S[%d] * %d\n", dst, src1, src2);;
      break;
    case 7: // DIV
      printf( "S[%d] = S[%d] / S[%d]\n", dst, src1, src2);;
      break;
    case 8: //DIVI
      printf( "S[%d] = S[%d] / %d\n", dst, src1, src2);;
      break;
    case 9:
      printf("NO LONGER SUPPORT SET");
      break;
    case 10:
      printf("NO LONGER SUPPORT POP");
      break;
    case 11:  // LSHIFT
      printf( "S[%d] = S[%d] << S[%d]\n", dst, src1, src2);;
      break;
    case 12:  // LSHIFTI 
      printf( "S[%d] = S[%d] << %d\n", dst, src1, src2);;
      break;   
    case 13:  // RSHIFT
      printf( "S[%d] = S[%d] >> S[%d]\n", dst, src1, src2);;
      break;
    case 14:  // RSHIFTI
      printf( "S[%d] = S[%d] >> %d\n", dst, src1, src2);;
      break;
    case 15:  //  OR
      printf( "S[%d] = S[%d] | S[%d]\n", dst, src1, src2);;
      break;
    case 16:  //  ORI
      printf( "S[%d] = S[%d] | %d\n", dst, src1, src2);;
      break;
    case 17:  //  AND
      printf( "S[%d] = S[%d] & S[%d]\n", dst, src1, src2);;
      break;
    case 18:  //  ANDI
      printf( "S[%d] = S[%d] & %d\n", dst, src1, src2);;
      break;
    case 0:
      printf("Epsilon\n");
      break;
      }
}



void FSTGPU::transition_convert ( uint32_t alphabet) {
int epsilon = 222;
  if ( alphabet < ALPHABETSIZE)  
    printf("%c",  alphabet);
  else if ( (alphabet >= ALPHABETSIZE)  && ( alphabet < EPSILON_MATCH) ){
    printf( "!%c", alphabet-ALPHABETSIZE);
    }
  else if (   alphabet  == EPSILON_MATCH) {
    printf("%c", epsilon);
  }
  else if ( ( alphabet  >= VARSTART ) && (alphabet  < (VARSTART + MAXVAR ) ) ) {
    printf( "S[%d]", alphabet - (VARSTART));
  }
  else if ( ( alphabet  >= INPUTSTART)  && (alphabet  < (INPUTSTART + MAXVAR) )  ) {
    printf("I[%d]",alphabet- (INPUTSTART));
  }
  else if (   alphabet  == ANY_MATCH) {
    printf("*");
  }
  else if ( ( alphabet  >= OUTPUTSTART) && (alphabet  < (OUTPUTSTART + MAXVAR) )  ){
    printf("O[%d]",alphabet- (OUTPUTSTART));
  }
  else if ( ( alphabet  >=  (VARSTART + ALPHABETSIZE) )  && ( alphabet < ( VARSTART + ALPHABETSIZE + MAXVAR)) ) {
    printf("!S[%d]", alphabet - (VARSTART) - (ALPHABETSIZE));
  }
  else if  ( (alphabet  >= (INPUTSTART + ALPHABETSIZE) ) && (alphabet < (INPUTSTART + MAXVAR + ALPHABETSIZE) )  ) {
    printf("!I[%d]", alphabet- (INPUTSTART) -(ALPHABETSIZE)) ;
  }
  else if (   alphabet  == ANY_PUSH) {
    printf("Push");
  }
}


