#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"
#include "processor.h"
#include "processor_gpu.h"

__global__ 
void printTape_D( uint32_t ** tape, uint32_t * size, uint32_t numberoftapes){
  for ( int i = 0; i < numberoftapes; i++){
    printf("Tape: %d : element: %d:\n", i, size[i]);
    for( int j = 0; j < size[i]; j++){
      printf("%d; ", tape[i][j]);
    }
    printf("\n");
  }
}

void printTape( uint32_t ** tape, uint32_t * size, uint32_t numberoftapes){
  for ( int i = 0; i < numberoftapes; i++){
    printf("Tape: %d : element: %d:\n", i, size[i]);
    for( int j = 0; j < size[i]; j++){
      printf("%d; ", tape[i][j]);
    }
    printf("\n");
  }
}
void FSTGPU::IO_setup( TP* cpu_transducer, uint32_t blocks, uint32_t threads){
  printf (" =====================================================\n");
  printf(" Start setting up input, output\n");
  number_of_thread = threads;
  number_of_block = blocks;
  total_threads = number_of_thread * number_of_block;
  printf("%d Input\n", cpu_transducer->inputCount);
  printf("%d Output\n", cpu_transducer->outputCount);
  inputCount = cpu_transducer->inputCount;
  outputCount = cpu_transducer->outputCount;
  printf("CPU INPUT\n");
  //printTape (&(cpu_transducer->inStream[0]), &(cpu_transducer->input_length[0]), inputCount); 

  hipError_t error;
  // copy input length array
  hipMalloc( &input_length , inputCount * sizeof (uint32_t));
  hipMemcpy ( input_length,cpu_transducer->input_length, inputCount * sizeof( uint32_t), hipMemcpyHostToDevice); 
 
  // allocate each input and copy them to device
  uint32_t ** temp_input = (uint32_t**) malloc ( inputCount *sizeof( uint32_t*));
  for ( uint32_t i = 0; i < inputCount; i++){
    error = hipMalloc ( &temp_input[i], cpu_transducer->input_length[i] * sizeof(uint32_t));
    //printf( "GPU MAlloc assert %s \n", hipGetErrorString(error));                                       
    error = hipMemcpy ( (temp_input[i]), (cpu_transducer->inStream[i]),cpu_transducer->input_length[i] * sizeof(uint32_t),  hipMemcpyHostToDevice);
    //printf( "GPU Copy assert %s \n", hipGetErrorString(error));                               
  }
  // copy input pointer to device
  hipMalloc( & (input), inputCount* sizeof( uint32_t*)); 
  hipMemcpy ((input), (temp_input), inputCount * sizeof( uint32_t*), hipMemcpyHostToDevice);

  //printf("GPU INPUT\n");
  //printTape_D<<< 1,1>>> (input, input_length, inputCount); 
  hipDeviceSynchronize();
  // allocate Output
#ifdef DEBUG_GPU
  printf(" ALLOCATE OUTPUT FOR DEBUG\n");
  uint32_t ** temp_output = (uint32_t**) malloc ( outputCount *sizeof( uint32_t*));
  // allocate each input and copy them to device
  for ( uint32_t i = 0; i < outputCount; i++){
    //hipMalloc ( &(temp_output[i]), OUTPUT_LENGTH * sizeof(uint32_t));
    hipMalloc ( &(temp_output[i]), cpu_transducer->output_length[i] * sizeof(uint32_t));
  }
  // copy input pointer to device
  hipMalloc( & (output), outputCount* sizeof( uint32_t*)); 
  hipMemcpy ((output), (temp_output), outputCount * sizeof( uint32_t*), hipMemcpyHostToDevice);
#endif
  printf (" =====================================================\n");
}


void FSTGPU::IO_partition(  TP * cpu_transducer, int test){

  printf (" =====================================================\n");
  printf(" START PARTITIONING IO\n");
// initialize partition data on CPU
  printf(" Allocating cpu for %d threads, %d input, %d output\n", total_threads, inputCount, outputCount);
  partition_input_length_cpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  partition_input_base_cpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  partition_input_current_cpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  partition_output_base_cpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  partition_output_current_cpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  for ( int i = 0; i < total_threads; i++){
    partition_input_length_cpu[i] = ( uint32_t *) malloc ( inputCount * sizeof(uint32_t));
    partition_input_base_cpu[i] = ( uint32_t *) malloc ( inputCount  * sizeof(uint32_t));
    partition_input_current_cpu[i] = ( uint32_t *) malloc ( inputCount  * sizeof(uint32_t));
    partition_output_base_cpu[i] = ( uint32_t *) malloc ( outputCount  * sizeof(uint32_t));
    partition_output_current_cpu[i] = ( uint32_t *) malloc ( outputCount  * sizeof(uint32_t));
  }
// parition according to the benchmark
  uint32_t * dummy_length = (uint32_t*) malloc ( inputCount * sizeof( uint32_t));
  hipMemcpy ( dummy_length, input_length, inputCount * sizeof(uint32_t), hipMemcpyDeviceToHost);
  //testing functionality
  printf(" Initializing cpu\n");
  partition(cpu_transducer,  test); 

  // allocate and copy partition data to GPU

  printf(" Allocating %d gpu  pointer \n", inputCount);
  uint32_t ** partition_input_length_gpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  uint32_t ** partition_input_base_gpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  uint32_t ** partition_input_current_gpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  uint32_t ** partition_output_base_gpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  uint32_t ** partition_output_current_gpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));
  uint32_t ** simulated_output_gpu  = (uint32_t **) malloc ( total_threads * sizeof( uint32_t *));


  printf(" Allocating gpu  \n");
  for ( uint32_t i = 0 ; i < total_threads ; i++){
    hipMalloc (&( partition_input_length_gpu[i]), inputCount * sizeof(uint32_t));
    hipMemcpy (partition_input_length_gpu[i], partition_input_length_cpu[i], inputCount * sizeof(uint32_t), hipMemcpyHostToDevice);

    hipMalloc (&( partition_input_base_gpu[i]), inputCount * sizeof(uint32_t));
    hipMemcpy (partition_input_base_gpu[i], partition_input_base_cpu[i], inputCount *sizeof(uint32_t), hipMemcpyHostToDevice);

    hipMalloc (&( partition_input_current_gpu[i]), inputCount * sizeof(uint32_t));
    hipMemcpy (partition_input_current_gpu[i], partition_input_current_cpu[i], inputCount *sizeof(uint32_t), hipMemcpyHostToDevice);
 
    hipMalloc (&( partition_output_base_gpu[i]), outputCount * sizeof(uint32_t));
    hipMemcpy (partition_output_base_gpu[i], partition_output_base_cpu[i], outputCount *sizeof(uint32_t), hipMemcpyHostToDevice);

    hipMalloc (&( partition_output_current_gpu[i]), outputCount * sizeof(uint32_t));
    hipMemcpy (partition_output_current_gpu[i], partition_output_current_cpu[i], outputCount *sizeof(uint32_t), hipMemcpyHostToDevice);

    hipMalloc (&( simulated_output_gpu[i]), outputCount * sizeof(uint32_t));
  }

  printf(" Copy GPU pointer \n");

  hipMalloc (&( partition_input_length),  total_threads * sizeof(uint32_t*));
  hipMemcpy (partition_input_length, partition_input_length_gpu, total_threads * sizeof(uint32_t*), hipMemcpyHostToDevice);

  hipMalloc (&( partition_input_base),  total_threads * sizeof(uint32_t*));
  hipMemcpy (partition_input_base, partition_input_base_gpu,  total_threads * sizeof(uint32_t*), hipMemcpyHostToDevice);

  hipMalloc (&( partition_input_current),  total_threads * sizeof(uint32_t*));
  hipMemcpy (partition_input_current, partition_input_current_gpu,  total_threads * sizeof(uint32_t*), hipMemcpyHostToDevice);

  hipMalloc (&( partition_output_base),  total_threads * sizeof(uint32_t*));
  hipMemcpy (partition_output_base, partition_output_base_gpu,  total_threads * sizeof(uint32_t*), hipMemcpyHostToDevice);

  hipMalloc (&( partition_output_current),  total_threads * sizeof(uint32_t*));
  hipMemcpy (partition_output_current, partition_output_current_gpu,  total_threads * sizeof(uint32_t*), hipMemcpyHostToDevice);

  hipMalloc (&( simulated_output), total_threads * sizeof(uint32_t*));
  hipMemcpy (simulated_output, simulated_output_gpu,  total_threads * sizeof(uint32_t*), hipMemcpyHostToDevice);
}


void FSTGPU::copyBack( TP * transducer){
  uint32_t ** tmp = (uint32_t **) malloc( outputCount * sizeof( uint32_t*));
  hipMemcpy ( tmp, output, outputCount * sizeof( uint32_t*), hipMemcpyDeviceToHost);
  for  (int i = 0; i <transducer-> outputCount; i++){
    hipMemcpy( transducer->outStream[i], tmp[i], OUTPUT_LENGTH * sizeof( uint32_t), hipMemcpyDeviceToHost);
    
  }
  #ifdef DEBUG_GPU
  for ( int i = 0; i < transducer->outputCount; i ++){
    printf("Output %d: \n", i);
    //for( int j =0; j < OUTPUT_LENGTH; j++){
    for( int j =0; j < 10; j++){
      printf( "%u; ", transducer->outStream[i][j]);
    }
    printf("\n");
  }
  #endif
}
