#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"

void FSTGPU::partition (TP * cpu_transducer, int test){
std::cout <<"=======================================" << std::endl;
  std::string destination  = "./testcases/";
  switch(test){
    case RLE_ENCODE:
      std::cout << " RLE_ENCODE" << std::endl;
      destination = destination +  "rle_encode/gpu_thread_config";
      RLE_ENCODING( cpu_transducer);

      break;
    case RLE_ENCODE_OPT:
      std::cout << " RLE_ENCODE_OPT" << std::endl;
      destination = destination +  "rle_encode_opt/gpu_thread_config";
      RLE_ENCODING( cpu_transducer);

      break;
    case RLE_DECODE:
      std::cout << " RLE_DECODE" << std::endl;
      destination = destination +  "rle_decode/gpu_thread_config";
      RLE_DECODING( cpu_transducer);

      break;
    case RLE_DECODE_OPT:
      std::cout << " RLE_DECODE_OPT" << std::endl;
      destination = destination +  "rle_decode_opt/gpu_thread_config";
      RLE_DECODING( cpu_transducer);
      break;
    case BP_ENCODE:
      std::cout << " BP_ENCODE" << std::endl;
      destination = destination +  "bp_encode/gpu_thread_config";
      BP_ENCODING( cpu_transducer);
      break;
    case BP_DECODE:
      std::cout << " BP_DECODE" << std::endl;
      destination = destination +  "bp_decode/gpu_thread_config";
      RLE_ENCODING( cpu_transducer);
      break;
    case GV_ENCODE:
      std::cout << " GV_ENCODE" << std::endl;
      destination = destination +  "gv_encode/gpu_thread_config";
      RLE_ENCODING( cpu_transducer);
      break;
    case GV_DECODE:
      std::cout << " GV_DECODE" << std::endl;
      destination = destination +  "gv_decode/gpu_thread_config";
      RLE_ENCODING( cpu_transducer);

      break;
    case DENSE_DOK:
      std::cout << " DENSE_DOK" << std::endl;

      break;

    case DOK_LIL:
      std::cout << " DOK_LIL" << std::endl;

      break;
    case LIL_COO:
      std::cout << " LIL_COO" << std::endl;

      break;
    case COO_CSR:
      std::cout << " COO_CSR" << std::endl;
      destination = destination +  "coo_csr/gpu_thread_config";
      COOCSR( cpu_transducer);

      break;
    case CSR_DENSE:
      std::cout << " CSR_DENSE" << std::endl;

      break;
    case DENSE_CSR_PTR:
      std::cout << " DENSE_CSR_PTR" << std::endl;
      destination = destination +  "dense_csr_ptr/gpu_thread_config";
      COOCSR( cpu_transducer);
      break;
    case CSV_ENC_DET:
      std::cout << " CSV_ENC_DET" << std::endl;
      destination = destination +  "csv_parsing/gpu_thread_config";
      CSV_PARSING(  cpu_transducer);
      break;
  }
  //save_thread_config(cpu_transducer, destination);
}
void FSTGPU::save_thread_config( TP * cpu_transducer, std::string destination){
  std::cout << "==================================== "<< std::endl;
  std::cout << "Saving thread config "<< std::endl;
  std::ofstream pconf  (destination, std::ofstream::out);
  pconf << number_of_block <<" Block | " << number_of_thread <<  " thread per block"<< std::endl;
  for ( uint32_t i = 0; i < total_threads; i++){
    pconf << "Thread " << i << std::endl;
    for (uint32_t j = 0; j < cpu_transducer->inputCount; j++){
      pconf<< "Input: " << j<< "| " ;
      pconf<< partition_input_base_cpu[i][j] << " - " << partition_input_length_cpu[i][j] << std::endl;
      pconf <<"\t" ;
      /*
      for ( uint32_t k = partition_input_base_cpu[i][j] ; k < partition_input_base_cpu[i][j] + partition_input_length_cpu[i][j] ; k++)
        pconf << cpu_transducer->inStream[j][k] << " ";
      pconf << std::endl;
      */
      pconf << "--------------------" << std::endl;
    }
    for (uint32_t j = 0; j < cpu_transducer->outputCount; j++){
      pconf<< "output: " << j<< "| " ;
      pconf<< partition_output_base_cpu[i][j] << " - " << partition_output_current_cpu[i][j] << std::endl;
      pconf <<"\t" ;
      pconf << "--------------------" << std::endl;
    }
  }
  pconf.close();
  std::cout << "==================================== "<< std::endl;
}


// RLE ENCODING
void FSTGPU::RLE_ENCODING( TP * cpu_transducer){
  std::cout << "==================================== "<< std::endl;
  std::cout << " START PARTITION INPUT FOR RLE, BPE "<< std::endl;
  uint32_t thread_length = cpu_transducer->input_length[0] / total_threads;

  if ( thread_length < MIN_LENGTH) thread_length = MIN_LENGTH;
  if ( cpu_transducer->input_length[0]  < MIN_LENGTH) thread_length = cpu_transducer->input_length[0];

  uint32_t max_thread = cpu_transducer->input_length[0]/thread_length;
  uint32_t left_over = cpu_transducer->input_length[0] % thread_length; 
  std::cout << " each thread process " << thread_length <<" symbols"<< std::endl;
  std::cout << " max thread " << max_thread <<" threads"<< std::endl;
  std::cout << " left over " << left_over <<" symbols"<< std::endl;

  for ( uint32_t i = 0; i < total_threads; i++){
    for (uint32_t j = 0; j < cpu_transducer->inputCount; j++){
      //std::cout << "thread : " << i << ", input: " << j << std::endl;
      if ( i < max_thread){
        partition_input_length_cpu[i][j] = thread_length; 
      }
      else if ( i == max_thread){ 
        partition_input_length_cpu[i][j] = left_over; 
      }
      else {
        partition_input_length_cpu[i][j] =0; 
      }
#ifdef SAME_START
      partition_input_base_cpu[i][j] = 0;  
#else
      partition_input_base_cpu[i][j] = i * thread_length;  
#endif
      partition_input_current_cpu[i][j] = 0;  
    }
    for (uint32_t j = 0; j < cpu_transducer->outputCount; j++){
      partition_output_base_cpu[i][j] = i * thread_length ;  
      partition_output_current_cpu[i][j] =0 ;  
    }
  }
  std::cout << "==================================== "<< std::endl;
}

// RLE DECODING
void FSTGPU::RLE_DECODING( TP * cpu_transducer){
  std::cout << "==================================== "<< std::endl;
  std::cout << " START PARTITION INPUT FOR RLD "<< std::endl;
  uint32_t thread_length = cpu_transducer->input_length[0] / total_threads;

  if ( thread_length < MIN_LENGTH) thread_length = MIN_LENGTH;
  if (( thread_length %2) != 0) thread_length++;
  if ( cpu_transducer->input_length[0]  < MIN_LENGTH) thread_length = cpu_transducer->input_length[0];

  uint32_t max_thread = cpu_transducer->input_length[0]/thread_length;
  uint32_t left_over = cpu_transducer->input_length[0] % thread_length; 
  if (( left_over %2) != 0) left_over= left_over-1;
  left_over= 0;
  std::cout << " each thread process " << thread_length <<" symbols"<< std::endl;
  std::cout << " max thread " << max_thread <<" threads"<< std::endl;
  std::cout << " left over " << left_over <<" symbols"<< std::endl;

  for ( uint32_t i = 0; i < total_threads; i++){
    for (uint32_t j = 0; j < cpu_transducer->inputCount; j++){
      //std::cout << "thread : " << i << ", input: " << j << std::endl;
      if ( i < max_thread){
        partition_input_length_cpu[i][j] = thread_length; 
      }
      else if ( i == max_thread){ 
        partition_input_length_cpu[i][j] = left_over; 
      }
      else {
        partition_input_length_cpu[i][j] =0; 
      }
#ifdef SAME_START
      partition_input_base_cpu[i][j] = 0;  
#else
      partition_input_base_cpu[i][j] = i * thread_length;  
#endif
      partition_input_current_cpu[i][j] = 0;  
    }
    for (uint32_t j = 0; j < cpu_transducer->outputCount; j++){
      partition_output_base_cpu[i][j] = i * thread_length ;  
      partition_output_current_cpu[i][j] =0 ;  
    }
  }
  std::cout << "==================================== "<< std::endl;
}

// BP_ENCODING
void FSTGPU::BP_ENCODING( TP * cpu_transducer){
  std::cout << "==================================== "<< std::endl;
  std::cout << " START PARTITION INPUT FOR RLD "<< std::endl;
  uint32_t thread_length = cpu_transducer->input_length[0] / total_threads;

  if ( thread_length < MIN_LENGTH) thread_length = MIN_LENGTH;
  if (( thread_length %4) != 0) thread_length = thread_length + ( 4- (thread_length%4));
  if ( cpu_transducer->input_length[0]  < MIN_LENGTH) thread_length = cpu_transducer->input_length[0];

  uint32_t max_thread = cpu_transducer->input_length[0]/thread_length;
  uint32_t left_over = cpu_transducer->input_length[0] % thread_length; 
  if (( left_over %4) != 0) left_over= left_over-1;
  left_over= 0;
  std::cout << " each thread process " << thread_length <<" symbols"<< std::endl;
  std::cout << " max thread " << max_thread <<" threads"<< std::endl;
  std::cout << " left over " << left_over <<" symbols"<< std::endl;

  for ( uint32_t i = 0; i < total_threads; i++){
    for (uint32_t j = 0; j < cpu_transducer->inputCount; j++){
      //std::cout << "thread : " << i << ", input: " << j << std::endl;
      if ( i < max_thread){
        partition_input_length_cpu[i][j] = thread_length; 
      }
      else if ( i == max_thread){ 
        partition_input_length_cpu[i][j] = left_over; 
      }
      else {
        partition_input_length_cpu[i][j] =0; 
      }
#ifdef SAME_START
      partition_input_base_cpu[i][j] = 0;  
#else
      partition_input_base_cpu[i][j] = i * thread_length;  
#endif
      partition_input_current_cpu[i][j] = 0;  
    }
    for (uint32_t j = 0; j < cpu_transducer->outputCount; j++){
      partition_output_base_cpu[i][j] = i * thread_length ;  
      partition_output_current_cpu[i][j] =0 ;  
    }
  }
  std::cout << "==================================== "<< std::endl;
}

// BP_ENCODING
void FSTGPU::COOCSR( TP * cpu_transducer){
  std::cout << "==================================== "<< std::endl;
  std::cout << " START PARTITION INPUT FOR COO-CSR  "<< std::endl;
  
  uint32_t number_of_data = cpu_transducer->inStream[0][2]; 
  uint32_t number_of_row = cpu_transducer->inStream[0][0]; 
  uint32_t number_of_col = cpu_transducer->inStream[0][1]; 
  std::cout << " Number of data: " << number_of_data <<std::endl; 
  std::cout << " Number of row: " << number_of_row <<std::endl; 
  std::cout << " Number of col: " << number_of_col <<std::endl; 
  /*
  uint32_t number_of_matrix = cpu_transducer->input_length[1] / number_of_data;
  uint32_t thread_length = number_of_matrix/ total_threads;
  uint32_t max_thread = cpu_transducer->input_length[1]/( thread_length* number_of_data);
  if ( max_thread <1) max_thread =1;
  uint32_t left_over = cpu_transducer->input_length[1] % (thread_length * number_of_data); 
  left_over= 0;
*/
  uint32_t thread_length  = cpu_transducer->input_length[1] / total_threads;
  if ( thread_length < 100) thread_length = 100;

  uint32_t max_thread = cpu_transducer->input_length[1]/( thread_length);
  if ( max_thread <1) max_thread =1;
  uint32_t left_over = cpu_transducer->input_length[1] % (thread_length); 
  left_over= 0;
  std::cout << " Input has  " << cpu_transducer->input_length[1] <<" symbol or  ";;
  //std::cout << number_of_matrix  <<" matrixes "<< std::endl;
  std::cout << " each thread process " << thread_length <<" matrix "<< std::endl;
  std::cout << " max thread " << max_thread <<" threads"<< std::endl;
  std::cout << " left over " << left_over <<" symbols"<< std::endl;

  for ( uint32_t i = 0; i < total_threads; i++){
    for (uint32_t j = 0; j < cpu_transducer->inputCount; j++){
      if ( j == 0){
        partition_input_length_cpu[i][j] =3; 
        partition_input_base_cpu[i][j] = 0;  
        partition_input_current_cpu[i][j] = 0;  
      }
      else{ 
        if ( i < max_thread){
          //partition_input_length_cpu[i][j] = thread_length * number_of_data; 
          partition_input_length_cpu[i][j] = thread_length ; 
        }
        else if ( i == max_thread){ 
          partition_input_length_cpu[i][j] = left_over; 
        }
        else {
          partition_input_length_cpu[i][j] =0; 
        }
        partition_input_base_cpu[i][j] = 0;  
        partition_input_current_cpu[i][j] = 0;  
      }
    }
    for (uint32_t j = 0; j < cpu_transducer->outputCount; j++){
      if ( j == 0) {
        partition_output_base_cpu[i][j] = i *3;  
        partition_output_current_cpu[i][j] =0 ;  
      }
      else{
        partition_output_base_cpu[i][j] = i * thread_length * number_of_data;  
        partition_output_current_cpu[i][j] =0 ;  
      }
    }
  }
  std::cout << "==================================== "<< std::endl;
}

// CSV PARSING
void FSTGPU::CSV_PARSING( TP * cpu_transducer){
  std::cout << "==================================== "<< std::endl;
  std::cout << " START PARTITION INPUT FOR RLE, BPE "<< std::endl;
  uint32_t thread_length = cpu_transducer->input_length[0] / total_threads;

  if ( thread_length < MIN_LENGTH) thread_length = MIN_LENGTH;
  if ( cpu_transducer->input_length[0]  < MIN_LENGTH) thread_length = cpu_transducer->input_length[0];

  uint32_t max_thread = cpu_transducer->input_length[0]/thread_length;
  uint32_t left_over = cpu_transducer->input_length[0] % thread_length; 
  std::cout << " each thread process " << thread_length <<" symbols"<< std::endl;
  std::cout << " max thread " << max_thread <<" threads"<< std::endl;
  std::cout << " left over " << left_over <<" symbols"<< std::endl;

  for ( uint32_t i = 0; i < total_threads; i++){
    for (uint32_t j = 0; j < cpu_transducer->inputCount; j++){
      //std::cout << "thread : " << i << ", input: " << j << std::endl;
      if ( i < max_thread){
        partition_input_length_cpu[i][j] = thread_length; 
      }
      else if ( i == max_thread){ 
        partition_input_length_cpu[i][j] = left_over; 
      }
      else {
        partition_input_length_cpu[i][j] =0; 
      }
#ifdef SAME_START
      partition_input_base_cpu[i][j] = 0;  
#else
      //partition_input_base_cpu[i][j] = i * thread_length;  
      partition_input_base_cpu[i][j] = 0;  
#endif
      partition_input_current_cpu[i][j] = 0;  
    }
    for (uint32_t j = 0; j < cpu_transducer->outputCount; j++){
      partition_output_base_cpu[i][j] = i * thread_length ;  
      partition_output_current_cpu[i][j] =0 ;  
    }
  }
  std::cout << "==================================== "<< std::endl;
}
