#include "hip/hip_runtime.h"
#include "processor_gpu.h"
#include "gpu_fst_engine.h"

__device__
void ExecuteAction_d_thread ( uint32_t * var, uint32_t src1, uint32_t src2, uint32_t dst, uint32_t opt, uint32_t startVar){
  uint32_t src1Value, src2Value;
  switch (opt){
    case ADD:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value + src2Value;
      break;
    case ADDI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value + src2Value;
      break;
    case SUB:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value - src2Value;
      break;
    case SUBI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value - src2Value;
      break;
    case MUL:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value * src2Value;
      break;
    case MULI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value * src2Value;
      break;
    case DIV:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value / src2Value;
      break;
    case DIVI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value / src2Value;
      break;
    case LSHIFT:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value << src2Value;
      break;
    case LSHIFTI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value << src2Value;
      break;
    case RSHIFT:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value >> src2Value;
      break;
    case RSHIFTI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value >> src2Value;
      break;
    case OR:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value | src2Value;
      break;
    case ORI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value | src2Value;
      break;
    case AND:
      src1Value = (var[src1 + startVar]);
      src2Value = (var[src2+ startVar]);
      var[dst + startVar] = src1Value & src2Value;
      break;
    case ANDI:
      src1Value = (var[src1 + startVar]);
      src2Value = src2;
      var[dst + startVar] = src1Value & src2Value;
      break;
    case EPSILON:
    default:
      break;
  }
}

__device__
void state_action_d_thread ( uint32_t * var, uint32_t  currentState, uint32_t startVar){
  uint32_t src1 = stateList[currentState].src1 ;
  uint32_t src2 = stateList[currentState].src2 ;
  uint32_t dst = stateList[currentState].dst ;
  uint32_t opt = stateList[currentState].opt;
  ExecuteAction_d_thread ( var, src1, src2, dst, opt, startVar);
}


__device__
uint32_t getInput_d_thread ( uint32_t *  input, uint32_t * processed_symbol, uint32_t * input_base, uint32_t inputID ){
  uint32_t base = input_base[inputID];
  uint32_t current =  processed_symbol[inputID];
  return input[base+  current];

}


__device__
bool compareSwitch_d_thread( uint32_t * var, uint32_t lhs, uint32_t rhs, uint32_t ** input, uint32_t * shared_input_base,  uint32_t *shared_processed_symbol,  uint32_t startVar){
  int condition = ILL;
  // alphabet match:  0-256
  if ( rhs < ALPHABETSIZE) {
    //printf(" alphabet match\n");
    condition = ALPHABET;
  }
  // negation alphabet  match: 256-511
  else if ( (rhs >= ALPHABETSIZE) && ( rhs < (2 * ALPHABETSIZE)) ) {
    condition = NEGALPHABET;
  }
  // EPSILON match: 512
  else if ( rhs == (2 * ALPHABETSIZE)) {
    condition = EPS;
  }
  // Variable match: 513-576
  else if (( rhs >  (2 * ALPHABETSIZE)) && ( rhs <= (2 * (ALPHABETSIZE) + MAXVAR))) {
    condition = VAR;
  }
  //  Input Match:  577-640
  else if( ( rhs >= INPUT_MATCH) && ( rhs <( INPUT_MATCH + MAXVAR))) {
    condition = INPT;
  }
  //  Passthrough Match:  641
  else if ( rhs == ANY_MATCH) {
    condition = PASSTHROUGH;
  }
  // negation variable match:  769-832
  else if (( rhs >  (3 * ALPHABETSIZE)) && ( rhs <= (3 * ALPHABETSIZE + MAXVAR))) {
    condition =NEGVAR;
  }
  // Negation Input Match:  833-896
  else if( ( rhs >=( INPUT_MATCH + ALPHABETSIZE)) && ( rhs <( INPUT_MATCH+ ALPHABETSIZE+ MAXVAR))) {
    condition = NEGINPT;
    }

  uint32_t varValue, inputValue;
  bool match = false;
  switch (condition)  {
    case  ALPHABET:
      if ( lhs == rhs) match = true;
      break;
    case  NEGALPHABET:
      if ( lhs  != (rhs- ALPHABETSIZE )) match = true;
      break;
    case  EPS:
      match = true;
      break;
    case  VAR:
      varValue = var[rhs- 2*ALPHABETSIZE + startVar -1 ];
      if ( lhs == varValue) match = true;
      break;
    case  INPT:
      inputValue = getInput_d_thread( input[rhs-INPUT_MATCH], shared_processed_symbol, shared_input_base,  rhs -INPUT_MATCH);
      if ( lhs == inputValue) match = true;
      break;
    case  PASSTHROUGH:
      match = true;
      break;
    case  NEGVAR:
      varValue = var[rhs- 3*ALPHABETSIZE -1 + startVar];
      if ( lhs != varValue) match = true;
      break;
    case  NEGINPT:
      inputValue = getInput_d_thread( input[rhs-INPUT_MATCH- ALPHABETSIZE], shared_processed_symbol, shared_input_base,  rhs - INPUT_MATCH - ALPHABETSIZE);
      if ( lhs != inputValue) match = true;
      break;
      }
    return match;

}


__device__
uint32_t transition_eval_d_thread( uint32_t * var, uint32_t  currentState,  uint32_t startVar, uint32_t **input, uint32_t * shared_input_base,   uint32_t *shared_processed_symbol){
  uint32_t baseID = stateList[currentState].baseID;
  uint32_t numberofTransition = stateList[currentState].numberofTransition;
  uint32_t returnID = TRANSITION_COUNT;
  for ( uint32_t i = baseID; i < (baseID + numberofTransition); i++){
    // input
    bool inputMatch = false;
    if ( transitionList[i].inputID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lhs = getInput_d_thread(input[ transitionList[i].inputID- INPUT_MATCH], shared_processed_symbol, shared_input_base,  transitionList[i].inputID- INPUT_MATCH);
      /*
      printf(" Transition %d :", i);
      printf(" left hand side = %d ;", lhs);
      printf(" right hand side = %d \n", transitionList[i].inputSymbol);
      */
      // call compare to right-hand side
      inputMatch = compareSwitch_d_thread ( var,  lhs,transitionList[i].inputSymbol, input, shared_input_base,  shared_processed_symbol, startVar);
    }
    else {
      inputMatch = true;
    }
  // variable
  bool varMatch = false;
    if ( transitionList[i].IvarID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lshVar = transitionList[i].IvarID + startVar - 2 *ALPHABETSIZE -1;
      uint32_t lhs = var[lshVar];
      // call compare to right-hand side
      varMatch = compareSwitch_d_thread ( var, lhs,transitionList[i].inputVar , input, shared_input_base, shared_processed_symbol, startVar);
    }
    else {
      varMatch = true;
    }

  if ( inputMatch && varMatch){
     return i;
    }
  }
  return returnID;
}


__device__
uint32_t translateOutput_d_thread ( uint32_t * var, uint32_t ** input, uint32_t * shared_input_base,  uint32_t * shared_processed_symbol,uint32_t outputVar, uint32_t startVar){
  uint32_t character = 1000; // illegal value

  if ( outputVar < ALPHABETSIZE)
    character = outputVar;
  else if (( outputVar >=ALPHABETSIZE) && ( outputVar <= 2*ALPHABETSIZE))
    return character;
  else if ( (outputVar > 2*(ALPHABETSIZE)) &&( outputVar < (2*ALPHABETSIZE + MAXVAR)) ){
    character = var[startVar + outputVar - 2*ALPHABETSIZE -1 ];

  }
  else if ( outputVar >= 2 * ALPHABETSIZE + MAXVAR +1){
    character = getInput_d_thread( input[outputVar- INPUT_MATCH], shared_processed_symbol,shared_input_base,  outputVar - INPUT_MATCH);
    //printf("Write to output %d: %d\n", outputVar-INPUT_MATCH, character);
  }
  return character;
}


__device__
uint32_t transition_write_d_thread(  uint32_t * var,  uint32_t chosenTransition, uint32_t startVar, uint32_t ** input, uint32_t * shared_input_base,  uint32_t*  processed_symbol, uint32_t ** output,uint32_t * simulated_output, uint32_t* outputCount , uint32_t *output_base){
  if( chosenTransition == TRANSITION_COUNT)
    return STATE_COUNT;

  // write to variable
  if ( transitionList[chosenTransition].OvarID != EPSILON_MATCH){
    uint32_t outputVar = startVar+ transitionList[chosenTransition].OvarID - 2 * ALPHABETSIZE - 1;
    //printf("Write Var %d \n", outputVar);
    var[outputVar]  =
      translateOutput_d_thread ( var,  input, shared_input_base,  processed_symbol, transitionList[chosenTransition].outputVar,startVar);
  }
  // write to output

  if ( transitionList[chosenTransition].outputID != EPSILON_MATCH){
    uint32_t output_dst = transitionList[chosenTransition].outputID - OUTPUT_START;
#ifdef DEBUG_GPU
    output[output_dst][output_base[output_dst] + outputCount[output_dst]]  =
#else
  simulated_output[output_dst]  =
#endif
      translateOutput_d_thread ( var, input, shared_input_base, processed_symbol, transitionList[chosenTransition].outputSymbol, startVar);
    outputCount[output_dst] ++;
  }

  //actiavate next State
  return transitionList[chosenTransition].nextState;
}


__device__
bool is_done_processed_thread ( uint32_t * current, uint32_t * expected){
  for ( int i = 0; i < INPUT; i++){
    if ( current[i] < expected[i])
      return false;
  }
  return true;
}


//////////////////
// main processing function
// partition_length: number of inputsymbol each stream have to processed
// input_base: element that each stream have to processed from
// processed_symbol: number of input each stream have processed
// partition_output_base: element each stream have to process from
// output_count: number of symbol have been written to each stream
__global__
void process_k_thread( TPGPU * fst,uint32_t ** input_base, uint32_t ** partition_length, uint32_t ** processed_symbol, uint32_t ** input, 
                             uint32_t ** partition_output_base, uint32_t** partition_output_count, uint32_t ** output, uint32_t **simulated_output ){
  uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t total_thread =   gridDim.x * blockDim.x ;
  uint32_t startVar = globalId * VARIABLE;
  uint32_t startVar_shared = threadIdx.x * VARIABLE;


  __shared__ uint32_t var [VARIABLE * THREAD_PER_BLOCK];
  __shared__ uint32_t shared_input_base[THREAD_PER_BLOCK][INPUT]; 
  __shared__ uint32_t shared_input_length[THREAD_PER_BLOCK][INPUT]; 
  __shared__ uint32_t shared_input_current[THREAD_PER_BLOCK][INPUT]; 
  //__shared__ uint32_t shared_output_current[THREAD_PER_BLOCK][OUTPUT];
// initialize shared variable
  for ( uint32_t i = 0; i < INPUT; i++){
    shared_input_base[threadIdx.x][i] = input_base[globalId][i];
    shared_input_length[threadIdx.x][i] = partition_length[globalId][i];
    shared_input_current[threadIdx.x][i] = 0;
  } 
  for ( uint32_t i = 0; i < VARIABLE; i++){
    var[threadIdx.x *VARIABLE + i] = 0 ;
  } 
  __syncthreads();
//printf(" Finish allocating shared variable\n");
#ifdef DEBUG_GPU
  uint32_t chosen_thread = 0;
  if (globalId == chosen_thread){
    printf( "State Table : %d state\n", STATE_COUNT);
    for ( uint32_t i = 0 ; i < STATE_COUNT; i++)      
      printf("S[%d]: %d, %d, %d, %d\n", stateList[i].id,  stateList[i].opt, stateList[i].src1, stateList[i].src2, stateList[i].dst);
    printf( "Transition Table : %d transition\n", TRANSITION_COUNT);
    for ( uint32_t i = 0 ; i < TRANSITION_COUNT; i++){      
      printf("T[%d]: %d-%d, %d-%d, %d | ", transitionList[i].id, transitionList[i].inputID, transitionList[i].inputSymbol, transitionList[i].IvarID, transitionList[i].inputVar, transitionList[i].currentState);          
      printf(" %d-%d, %d-%d, %d \n", transitionList[i].outputID, transitionList[i].outputSymbol, transitionList[i].OvarID, transitionList[i].outputVar, transitionList[i].nextState);
    }
    printf(" Allocate %d variable ", VARIABLE * THREAD_PER_BLOCK + 2*THREAD_PER_BLOCK*INPUT);
    printf(" of %d byte \n",sizeof(uint32_t)*  ( VARIABLE * THREAD_PER_BLOCK + 2*THREAD_PER_BLOCK*INPUT));
    for (uint32_t i = startVar_shared ; i < ( startVar_shared +  VARIABLE ) ; i++){
      printf( "V[%d]=%d , ", i , var[i]);
    }
    printf("\n");
      for (uint32_t j = 0; j < INPUT; j++){
        printf( "Input_length[%d][%d]=%d , ",threadIdx.x, j, shared_input_length[threadIdx.x][j]);
        printf( "Input_current[%d][%d]=%d \n ",threadIdx.x, j, shared_input_current[threadIdx.x][j]);
      }
    printf("\n");
  }
#endif
// execution
  
  uint32_t currentState = 0;
  uint32_t stateCount = STATE_COUNT;
  bool done = false;
  uint32_t chosenTransition = TRANSITION_COUNT;
  uint32_t cycle = 0;
#ifdef DEBUG_GPU
  while ( cycle < 200){
    __syncthreads();
    if ( globalId == chosen_thread){
      printf("-------------%d-----------------\n", cycle);
      printf( " Activated State: %d\n", currentState);
      for (uint32_t v = startVar; v < ( startVar_shared+  VARIABLE ) ; v++)
        printf( "V[%d] = %u | ", v, var[v]);
      printf("\n");

      for (uint32_t p = 0; p < INPUT; p++){
        printf("Input[%d]: ", p);
      for (uint32_t q = shared_input_base[threadIdx.x][p]; q <(shared_input_base[threadIdx.x][p] +   10); q++)
        printf("%d, ", input[p][q]);
      printf("\n");

      printf("Current[%d]:  %d - %d total %d \n",p,shared_input_base[threadIdx.x][p],  shared_input_current[threadIdx.x][p], shared_input_length[threadIdx.x][p]);
      for (uint32_t k = shared_input_base[threadIdx.x][p]; k <(shared_input_base[globalId][p] +  shared_input_current[threadIdx.x][p]); k++)
        printf("%u, ", input[p][k]);
      printf("\n");
      }
      for (uint32_t p = 0; p < OUTPUT; p++){
        printf("OutPut[%d]: ", p);
        for (uint32_t k = partition_output_base[globalId][p]; k < partition_output_base[globalId][p] + partition_output_count[globalId][p]; k++){
          printf("%u, ", output[p][k]);
        }
        printf("\n");
      }
    printf("Rerform Action\n");
    }
    cycle++;
#else
  while (  ( !done) && ( currentState != stateCount)){
#endif

    // state action
    state_action_d_thread( var, currentState, threadIdx.x * VARIABLE);
    
#ifdef DEBUG_GPU
    if ( globalId == chosen_thread)   printf( "Perform Action on state %d\n", currentState);
#endif
    // transition evaluation
    chosenTransition = transition_eval_d_thread( var, currentState, startVar_shared,  input, shared_input_base[threadIdx.x],  shared_input_current[threadIdx.x]);
     //__syncthreads();
#ifdef DEBUG_GPU
    if ( globalId == chosen_thread) {
      uint32_t baseID = stateList[currentState].baseID;
      uint32_t numberofTransition = stateList[currentState].numberofTransition;
      printf( "Evaluate Transition on T%d\n", chosenTransition);
      printf(" Considering Transition %d to %d\n", baseID, baseID+ numberofTransition -1 );
    }
#endif
    currentState =
    transition_write_d_thread(var, chosenTransition, startVar_shared,input, shared_input_base[threadIdx.x],   shared_input_current[threadIdx.x],  output,simulated_output[globalId],  partition_output_count[globalId], partition_output_base[globalId]);

     //__syncthreads();
#ifdef DEBUG_GPU
    if ( globalId == chosen_thread) printf( "Write Next State %d\n", currentState);
#endif

    // consumed input and deactivate transition
    if (currentState == STATE_COUNT){
      done = true;
    }
    else {
      if( transitionList[chosenTransition].inputID != EPSILON_MATCH) {
        shared_input_current[threadIdx.x][transitionList[chosenTransition].inputID-INPUT_MATCH]++;
        chosenTransition = TRANSITION_COUNT;
        }
    done = is_done_processed_thread( shared_input_current[threadIdx.x], shared_input_length[threadIdx.x]);
    }
  }
  __syncthreads();
#ifdef DEBUG_GPU
  if ( globalId == chosen_thread){
    printf("Thread %d finished with %d cycle, %d output \n",  globalId, cycle, partition_output_count[globalId][0]);
    printf( " total Thread %d \n",  gridDim.x * blockDim.x );
    for ( uint32_t j = 0; j < fst->inputCount; j++){
      printf("T%d:  %d, ",globalId,  partition_output_base[globalId][j]);
      printf(" %d|  ", partition_output_count[globalId][j]);
    }
  }
#endif
}

void FSTGPU::process_thread( TPGPU * transducer){
  printf("Begin Execution\n");
  printf(" Number of block: %d \n", number_of_block);
  printf(" Number of thread per block: %d\n", number_of_thread);
  hipEvent_t start_execution, stop_execution;
  hipEventCreate( &start_execution);
  hipEventCreate( &stop_execution);

  hipEventRecord( start_execution, 0);
  process_k_thread <<< number_of_block,number_of_thread>>> (transducer,partition_input_base,  partition_input_length, partition_input_current, input,  partition_output_base,  partition_output_current,  output , simulated_output);
  hipEventRecord( stop_execution,0);
  hipEventSynchronize(stop_execution);
  float execution_time_ms = 0;
  hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
  //hipDeviceSynchronize();
  printf("Execution  Finished\n");
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess )
    printf("CUDA Error: %s\n", hipGetErrorString(err));       
  fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}

void FSTGPU::topo_global_to_constant_thread(TP * cpu_transducer){

  hipMemcpyToSymbol(HIP_SYMBOL( stateList), cpu_transducer->stateList, STATE_COUNT * sizeof(NPDT));
  hipMemcpyToSymbol(HIP_SYMBOL( transitionList), cpu_transducer->transitionList, TRANSITION_COUNT * sizeof(Transition));
}

