#include "hip/hip_runtime.h"
#include "processor_gpu.h"
#include "gpu_fst_engine.h"
__device__
void ExecuteAction_d ( TPGPU * fst, uint32_t src1, uint32_t src2, uint32_t dst, uint32_t opt, uint32_t startVar){
  uint32_t src1Value, src2Value;
  switch (opt){
    case ADD: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value + src2Value;
      break;
    case ADDI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value + src2Value; 
      break;
    case SUB: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value - src2Value;
      break;
    case SUBI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value - src2Value; 
      break;
    case MUL: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value * src2Value;
      break;
    case MULI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value * src2Value; 
      break;
    case DIV: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value / src2Value;
      break;
    case DIVI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value / src2Value; 
      break;
    case LSHIFT: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value << src2Value;
      break;
    case LSHIFTI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value << src2Value; 
      break;
    case RSHIFT: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value >> src2Value;
      break;
    case RSHIFTI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value >> src2Value; 
      break;
    case OR: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value | src2Value;
      break;
    case ORI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value | src2Value; 
      break;
    case AND: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = (fst->var[src2+ startVar]); 
      fst->var[dst + startVar] = src1Value & src2Value;
      break;
    case ANDI: 
      src1Value = (fst->var[src1 + startVar]); 
      src2Value = src2; 
      fst->var[dst + startVar] = src1Value & src2Value; 
      break;
    case EPSILON:
    default:
      break;
  }
}


__device__
void state_action_d ( TPGPU * fst, uint32_t  currentState, uint32_t startVar){
  uint32_t src1 = fst->stateList[currentState].src1 ;
  uint32_t src2 = fst->stateList[currentState].src2 ;
  uint32_t dst = fst->stateList[currentState].dst ;
  uint32_t opt = fst->stateList[currentState].opt;
  ExecuteAction_d ( fst, src1, src2, dst, opt, startVar);
}

__device__ 
uint32_t getInput_d ( uint32_t *  input, uint32_t * processed_symbol, uint32_t * input_base, uint32_t inputID ){
    uint32_t base = input_base[inputID] ;
  uint32_t current =  processed_symbol[inputID];
  uint32_t current_input = input[base  + current];
  //uint32_t current_input = 99;
  return current_input;

}

__device__
bool compareSwitch_d( TPGPU* fst, uint32_t lhs, uint32_t rhs, uint32_t ** input, uint32_t *processed_symbol, uint32_t * input_base, uint32_t startVar){
  int condition = ILL;
  // alphabet match:  0-256
  if ( rhs < ALPHABETSIZE) {
    //printf(" alphabet match\n");
    condition = ALPHABET;
  }
  // negation alphabet  match: 256-511
  else if ( (rhs >= ALPHABETSIZE) && ( rhs < (2 * ALPHABETSIZE)) ) {
    condition = NEGALPHABET;
  }
  // EPSILON match: 512
  else if ( rhs == (2 * ALPHABETSIZE)) {
    condition = EPS;
  }
  // Variable match: 513-576
  else if (( rhs >  (2 * ALPHABETSIZE)) && ( rhs <= (2 * (ALPHABETSIZE) + MAXVAR))) {
    condition = VAR;
  } 
  //  Input Match:  577-640      
  else if( ( rhs >= INPUT_MATCH) && ( rhs <( INPUT_MATCH + MAXVAR))) {
    condition = INPT;
  }
  //  Passthrough Match:  641    
  else if ( rhs == ANY_MATCH) {   
    condition = PASSTHROUGH;
  }
  // negation variable match:  769-832
  else if (( rhs >  (3 * ALPHABETSIZE)) && ( rhs <= (3 * ALPHABETSIZE + MAXVAR))) {
    condition =NEGVAR;
  }
  // Negation Input Match:  833-896
  else if( ( rhs >=( INPUT_MATCH + ALPHABETSIZE)) && ( rhs <( INPUT_MATCH+ ALPHABETSIZE+ MAXVAR))) {     
    condition = NEGINPT;
    }

  uint32_t varValue, inputValue;
  bool match = false;
  switch (condition)  {
    case  ALPHABET:
      if ( lhs == rhs) match = true;
      break;
    case  NEGALPHABET:
      if ( lhs  != (rhs- ALPHABETSIZE )) match = true;
      break;
    case  EPS:
      match = true;
      break;
    case  VAR:
      varValue = fst->var[rhs- 2*ALPHABETSIZE + startVar -1 ];
      if ( lhs == varValue) match = true;
      break;
    case  INPT:
      inputValue = getInput_d( input[rhs-INPUT_MATCH], processed_symbol, input_base, rhs -INPUT_MATCH);
      if ( lhs == inputValue) match = true;
      break;
    case  PASSTHROUGH:
      match = true;
      break;
    case  NEGVAR:
      varValue = fst->var[rhs- 3*ALPHABETSIZE -1 + startVar];
      if ( lhs != varValue) match = true;
      break;
    case  NEGINPT:
      inputValue = getInput_d( input[rhs-INPUT_MATCH- ALPHABETSIZE], processed_symbol, input_base, rhs - INPUT_MATCH - ALPHABETSIZE);
      if ( lhs != inputValue) match = true;
      break;
      }
    return match;

}
__device__
uint32_t transition_eval_d(TPGPU * fst, uint32_t  currentState,  uint32_t startVar, uint32_t **input, uint32_t *input_base, uint32_t *processed_symbol){
  uint32_t baseID = fst->stateList[currentState].baseID;
  uint32_t numberofTransition = fst->stateList[currentState].numberofTransition;
  uint32_t returnID = fst->transitionCount;
  for ( uint32_t i = baseID; i < (baseID + numberofTransition); i++){
    // input
    bool inputMatch = false;
    if ( fst->transitionList[i].inputID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lhs = getInput_d(input[ fst->transitionList[i].inputID- INPUT_MATCH], processed_symbol, input_base, fst->transitionList[i].inputID- INPUT_MATCH);
      /*
      printf(" Transition %d :", i);
      printf(" left hand side = %d ;", lhs);
      printf(" right hand side = %d \n", fst->transitionList[i].inputSymbol);
      */
      // call compare to right-hand side
      inputMatch = compareSwitch_d ( fst, lhs,fst->transitionList[i].inputSymbol, input, processed_symbol, input_base, startVar);
    }
    else {
      inputMatch = true;
    }
  // variable
  bool varMatch = false;
    if ( fst->transitionList[i].IvarID != EPSILON_MATCH){
      // decode left-hand side
      uint32_t lshVar = fst->transitionList[i].IvarID + startVar - 2 *ALPHABETSIZE -1;
      uint32_t lhs = fst->var[lshVar];
      // call compare to right-hand side
      varMatch = compareSwitch_d ( fst, lhs,fst->transitionList[i].inputVar , input, processed_symbol, input_base, startVar);
    }
    else {
      varMatch = true;
    }

  if ( inputMatch && varMatch){
      //printf(" Match transition %d\n", i);
     return i;
     //returnID = i;
    }
  }
  return returnID;
}

__device__
uint32_t translateOutput_d ( TPGPU * fst, uint32_t ** input, uint32_t * processed_symbol,uint32_t * input_base,  uint32_t outputVar, uint32_t startVar){
  uint32_t character = 1000; // illegal value

  if ( outputVar < ALPHABETSIZE) 
    character = outputVar;
  else if (( outputVar >=ALPHABETSIZE) && ( outputVar <= 2*ALPHABETSIZE))
    return character;
  else if ( (outputVar > 2*(ALPHABETSIZE)) &&( outputVar < (2*ALPHABETSIZE + MAXVAR)) ){
    character = fst->var[startVar + outputVar - 2*ALPHABETSIZE -1 ];

  }
  else if ( outputVar >= 2 * ALPHABETSIZE + MAXVAR +1){
    character = getInput_d( input[outputVar- INPUT_MATCH], processed_symbol, input_base, outputVar - INPUT_MATCH);
    //printf("Write to output %d: %d\n", outputVar-INPUT_MATCH, character);
  }
  return character;
}


__device__
uint32_t transition_write_d( TPGPU * fst, uint32_t chosenTransition, uint32_t startVar, uint32_t ** input,uint32_t * input_base,  uint32_t*  processed_symbol, uint32_t ** output,uint32_t * simulated_output, uint32_t* outputCount , uint32_t *output_base){
  if( chosenTransition == fst->transitionCount)
    return fst->stateCount;

  // write to variable
  if ( fst->transitionList[chosenTransition].OvarID != EPSILON_MATCH){
    uint32_t outputVar = startVar+ fst->transitionList[chosenTransition].OvarID - 2 * ALPHABETSIZE - 1; 
    //printf("Write Var %d \n", outputVar);
    fst->var[outputVar]  =  
      translateOutput_d ( fst, input, processed_symbol,input_base, fst->transitionList[chosenTransition].outputVar,startVar);  
  }
  // write to output
  
  if ( fst->transitionList[chosenTransition].outputID != EPSILON_MATCH){
    uint32_t output_dst = fst->transitionList[chosenTransition].outputID - OUTPUT_START;
#ifdef DEBUG_GPU
    output[output_dst][output_base[output_dst] + outputCount[output_dst]]  = 
#else
  simulated_output[output_dst]  = 
 // uint32_t tmp =  
#endif
    //output[0][0]  = 
    //0;
      translateOutput_d ( fst, input, processed_symbol,input_base, fst->transitionList[chosenTransition].outputSymbol, startVar);  
    outputCount[output_dst] ++;
   // printf("AFTER Write output  %d \n", tmp);
    //printf(" From %d ", output_base[output_dst]);
    //printf("To: %d\n",outputCount[output_dst]); 

  }
 
  //actiavate next State
  return fst->transitionList[chosenTransition].nextState;
}

__device__
bool is_done_processed ( uint32_t * current, uint32_t * expected, uint32_t number_of_input){
  for ( int i = 0; i < number_of_input; i++){
    //printf(" Current %d: %d| Expected %d\n", i,  current[i] , expected[i]);
    if ( current[i] < expected[i])
      return false;
  }
  return true;
}
__device__
int testk ( uint32_t  testarray){
  return ++testarray;
}
//////////////////
// main processing function
// partition_length: number of inputsymbol each stream have to processed
// input_base: element that each stream have to processed from
// processed_symbol: number of input each stream have processed
// partition_output_base: element each stream have to process from
// output_count: number of symbol have been written to each stream
__global__
void process_k( TPGPU * fst, uint32_t ** input_base, uint32_t ** partition_length, uint32_t ** processed_symbol, uint32_t ** input, 
                             uint32_t ** partition_output_base, uint32_t** partition_output_count, uint32_t ** output, uint32_t **simulated_output ){
  uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t total_thread =   gridDim.x * blockDim.x ;
  uint32_t startVar = globalId * fst->varCount;
  uint32_t currentState = 0;
  uint32_t chosenTransition = fst->transitionCount;
  bool done = false;
  uint32_t cycle= 0;
  uint32_t t = 0;
#ifdef DEBUG_GPU
  uint32_t chosen_thread =0;
  if ( globalId == chosen_thread){
    for ( uint32_t i = 0; i < fst->inputCount; i++){
      printf(" %d, ", input_base[globalId][i]);
      printf(" %d| ", partition_length[globalId][i]);
    }
    for ( uint32_t i = 0; i < fst->outputCount; i++){
      printf(" %d, ", partition_output_base[globalId][i]);
      printf(" %d, ", partition_output_count[globalId][i]);
    }
    printf("\nSTART EXECUTION on var %d\n", startVar);
  }
#endif
#ifdef DEBUG_GPU
  while ( cycle < 20){
  
    cycle++;
    __syncthreads();
    if ( globalId == chosen_thread){
      printf("-------------%d-----------------\n", cycle);
      printf( " Activated State: %d\n", currentState);
      for ( uint32_t v = startVar; v < startVar + fst->varCount; v++)
        printf( "V[%d] = %u | ", v, fst->var[v]); 
      printf("\n");

      for (uint32_t p = 0; p < fst->inputCount; p++){
        printf("Input[%d]: ", p);
        for (uint32_t q = input_base[globalId][p]; q < input_base[globalId][p] + 20; q++){
          printf("%d, ", input[p][q]);
        }
      
        printf("\n");
        printf("Current[%d]:  %d - %d total %d \n",p,  input_base[globalId][p], processed_symbol[globalId][p], partition_length[globalId][p]);
        for (uint32_t k = input_base[globalId][p]; k <= input_base[globalId][p] + processed_symbol[globalId][p]; k++){
          printf("%u, ", input[p][k]);
        }
        printf("\n");
      }
      for (uint32_t p = 0; p < fst->outputCount; p++){
        printf("OutPut[%d]: ", p);
        for (uint32_t k = partition_output_base[globalId][p]; k < partition_output_base[globalId][p] + partition_output_count[globalId][p]; k++){
          printf("%u, ", output[p][k]);
        }
        printf("\n");
      }
    }
    __syncthreads();
    //printf("Rerform Action\n");
#else    
  while (( !done) && (currentState != fst->stateCount)){
#endif
    // state action
    state_action_d ( fst, currentState, startVar);
    // transition evaluate
    uint32_t baseID = fst->stateList[currentState].baseID;
    uint32_t numberofTransition = fst->stateList[currentState].numberofTransition;
    //printf(" Considering Transition %d to %d\n", baseID, baseID+ numberofTransition -1 );
    chosenTransition = transition_eval_d( fst, currentState, startVar,  input, input_base[globalId], processed_symbol[globalId]);
    // transition write
#ifdef DEBUG_GPU
    if ( globalId == chosen_thread){
      printf("Chosen Transition %d \n", chosenTransition);
      printf("%d-%d, %d-%d, %d |", fst->transitionList[chosenTransition].inputID, fst->transitionList[chosenTransition].inputSymbol, fst->transitionList[chosenTransition].IvarID,fst->transitionList[chosenTransition].inputVar, fst->transitionList[chosenTransition].currentState);   
      printf("%d-%d, %d-%d, %d \n", fst->transitionList[chosenTransition].outputID, fst->transitionList[chosenTransition].outputSymbol, fst->transitionList[chosenTransition].OvarID,fst->transitionList[chosenTransition].outputVar, fst->transitionList[chosenTransition].nextState); 
      printf( "%d\n" ,startVar+ fst->transitionList[chosenTransition].OvarID - 2 * ALPHABETSIZE - 1); 
      printf("Writing from output %d :  %d to %d\n", fst->transitionList[chosenTransition].outputID - OUTPUT_START, partition_output_base[globalId][0], partition_output_count[globalId][0]);
    }
    __syncthreads();
#endif
  currentState = 
    transition_write_d(fst, chosenTransition, startVar,input,input_base[globalId],  processed_symbol[globalId],  output,simulated_output[globalId],  partition_output_count[globalId], partition_output_base[globalId]);
  #ifdef DEBUG_GPU
    __syncthreads();
    if ( globalId == chosen_thread)
      printf( " Next State = %d\n", currentState);
  #endif
    // consumed input and deactivate transition
    if (currentState == fst->stateCount){
      done = true;
    }
    else {
      if( fst->transitionList[chosenTransition].inputID != EPSILON_MATCH) {
      //  printf(" Increament Input  %d from %d\n", fst->transitionList[chosenTransition].inputID-INPUT_MATCH, processed_symbol[globalId][fst->transitionList[chosenTransition].inputID-INPUT_MATCH]);
        processed_symbol[globalId][fst->transitionList[chosenTransition].inputID-INPUT_MATCH]++;
       // printf(" to %d\n", processed_symbol[globalId][fst->transitionList[chosenTransition].inputID-INPUT_MATCH]);
        chosenTransition = fst->transitionCount;
        }
//        printf("T: %d : \n", globalId);;
    done = is_done_processed( processed_symbol[globalId], partition_length[globalId], fst->inputCount);
    }
   // printf( "done: %d, current State : %d\n", done, currentState);

  }
/*
  if ( globalId == 0)
    printf("Thread %d finished with %d cycle, %d output \n",  globalId, cycle, partition_output_count[globalId][0]);
*/
#ifdef DEBUG_GPU
  if ( globalId == chosen_thread){
    printf("Thread %d finished with %d cycle, %d output \n",  globalId, cycle, partition_output_count[globalId][0]);
    printf( " total Thread %d \n",  gridDim.x * blockDim.x );
    for ( uint32_t j = 0; j < fst->inputCount; j++){
      printf("T%d:  %d, ",globalId, partition_output_base[globalId][j]);
      printf(" %d|  ", partition_output_count[globalId][j]);
    }
  }
#endif
}

void FSTGPU::process( TPGPU * transducer){
  printf("Begin Execution\n");
  printf(" Number of block: %d \n", number_of_block);
  printf(" Number of thread per block: %d\n", number_of_thread);
  hipEvent_t start_execution, stop_execution;
  hipEventCreate( &start_execution);
  hipEventCreate( &stop_execution);
  hipEventRecord( start_execution, 0);
  process_k <<< number_of_block,number_of_thread>>> (transducer, partition_input_base, partition_input_length, partition_input_current, input,  partition_output_base,  partition_output_current,  output , simulated_output);
  hipEventRecord( stop_execution,0);
  hipEventSynchronize(stop_execution);
  float execution_time_ms = 0;
  hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
  //hipDeviceSynchronize();
  printf("Execution  Finished\n");
  fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}
